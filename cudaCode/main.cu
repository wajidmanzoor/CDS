#include "./inc/graph.h"
#include "./inc/motif.h"

#include "./src/gpuMemoryAllocation.cc"
#include "./src/helpers.cc"


#define CUDA_CHECK_ERROR(kernelName) { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA Error in kernel %s, file %s at line %d: %s\n", kernelName, __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

bool fileExists(const string& filename) {
    struct stat buffer;
    return (stat(filename.c_str(), &buffer) == 0);
}

void writeOrAppend(const string& filename, const string& data) {
    ofstream file;
    
    // Check if the file exists
    if (fileExists(filename)) {
        // Open the file in append mode if it exists
        file.open(filename, ios::app);
    } else {
        // Open the file in write mode if it doesn't exist
        file.open(filename);
    }
    
    if (file.is_open()) {
        file << data << endl;
        file.close();
    } else {
        cerr << "Unable to open the file." << endl;
    }
}

int main(int argc, const char * argv[]) {
    if (argc != 6) {
        cout << "Server wrong input parameters!" << endl;
        exit(1);
    }

    string filepath = argv[1]; // Path to the graph file. The graph should be represented as an adjacency list with space separators
    string motifPath = argv[2]; //Path to motif file. The motif should be represented as edge list with space sperators.
    ui k = atoi(argv[3]);
    ui pSize = atoi(argv[4]);
    ui cpSize = atoi(argv[5]);

    Graph G = Graph(filepath);

    Motif M = Motif(motifPath);

    vector<ui> *listingOrder;
    
    G.getListingOrder(&listingOrder);
    memoryAllocationGraph(deviceGraph,G);
    memoryAllocationDAG(deviceDAG,G.n;G.m);

    // Get out degree in DAG
    generateDegreeDAG<<<BLK_NUMS, BLK_DIM>>>(deviceGraph,deviceDAG,listingOrder,G.n, G.m,TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree of DAG");

    //copy out degree to offset
    chkerr(hipMemset(deviceDAG.neighbor, 0, sizeof(ui)));
    chkerr(hipMemcpy(deviceDAG.offset + 1,deviceDAG.degree , (G.n) * sizeof(ui), cudaMemcpyDevicetoDevice));

    // cummulative sum offset
    thrust::inclusive_scan(thrust::device_ptr < ui > (deviceDAG.offset), thrust::device_ptr < ui > (deviceDAG.offset + G.n + 1), thrust::device_ptr < ui > (deviceDAG.offset ));

    // Write neighbors of DAG
    generateNeighborDAG<<<BLK_NUMS, BLK_DIM>>>(deviceGraph,deviceDAG,listingOrder,G.n, G.m,TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbors of DAG");
    int maxDegree =0;
    ui maxBitMask = memoryAllocationlevelData(levelData,k,pSize,cpSize,maxDegree,TOTAL_WARPS);
    int level =0;
    int iterK = k;

    ui *labels;
    chkerr(hipMalloc((void**)&(labels), n * sizeof(ui)));

    thrust::device_ptr<ui> dev_labels(labels);
    thrust::fill(dev_labels, dev_labels + n, iterK);

    chkerr(hipMemcpy(G.degree, graph.degree->data(), n * sizeof(ui), hipMemcpyHostToDevice));
    //TODO:  Declare Labels and fill with iterK 
    //TODO SHARED MEMORY 
    listIntialCliques<<<BLK_NUMS, BLK_DIM>>>(deviceDAG, levelData, label,iterK, G.n, G.m,psize,cpSize,maxBitMask,level,TOTAL_WARPS);
    iterK --;
    level ++;
    ui offsetPartitionSize = ((psize/(k-1)) + 1);
    thrust::transform(thrust::device, thrust::make_counting_iterator(0), thrust::make_counting_iterator(TOTAL_WARPS), leveldata.temp + 1,
                      [leveldata.offsetPartition, leveldata.count,offsetPartitionSize] __device__ (int i) {
                          int task_count = leveldata.count[i];
                          return (task_count > 0) ? leveldata.offsetPartition[i * offsetPartitionSize + task_count] : 0;
                      });

    thrust::inclusive_scan(thrust::device, leveldata.temp, leveldata.temp + TOTAL_WARPS + 1, leveldata.temp);
    thrust::inclusive_scan(thrust::device, leveldata.count, leveldata.count + TOTAL_WARPS + 1, leveldata.count);

    flushParitions<<<BLK_NUMS, BLK_DIM>>>( D, levelData,psize,cpSize,maxBitMask, level,TOTAL_WARPS);
    int totalTasks;
    chkerr(hipMemcpy(&totalTasks,leveldata.count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));

    
    while(iterK > 2 ){

        //TODO: need to think a solution for label . Maybe use it masks will see 
        thrust::fill(dev_labels, dev_labels + n, iterK);
        listMidCliques(D, levelData,label,k,iterK, G.n, G.m,psize,cpSize,maxBitMask,totalTasks,level,TOTAL_WARPS);
        thrust::transform(thrust::device, thrust::make_counting_iterator(0), thrust::make_counting_iterator(TOTAL_WARPS), leveldata.temp + 1,
                      [leveldata.offsetPartition, leveldata.count,offsetPartitionSize] __device__ (int i) {
                          int task_count = leveldata.count[i];
                          return (task_count > 0) ? leveldata.offsetPartition[i * offsetPartitionSize + task_count] : 0;
                      });

        thrust::inclusive_scan(thrust::device, leveldata.temp, leveldata.temp + TOTAL_WARPS + 1, leveldata.temp);
        thrust::inclusive_scan(thrust::device, leveldata.count, leveldata.count + TOTAL_WARPS + 1, leveldata.count);

        flushParitions<<<BLK_NUMS, BLK_DIM>>>( D, levelData,psize,cpSize,k,maxBitMask,level,TOTAL_WARPS);
        chkerr(hipMemcpy(&totalTasks,leveldata.count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));
        iterK --; 
        level ++;
    }
    if(iterK == 2){
        

    }


    //TODO: Free labels, 
    //TODO: try to divide the Partition data and actual data into two data structures, so we can free partition before declaring trie
    // A way to calculate T from num partial cliques X candidate sets
    ui t; 
    memoryAllocationTrie(cliques, t, k);
    
    //call k==2 kernel 



     
    freeGraph(deviceGraph);
    freeDAG(deviceDAG);
    delete G;
    delete M;
    return 0;
}
