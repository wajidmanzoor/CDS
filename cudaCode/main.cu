#include "./inc/common.h"
#include "./inc/graph.h"
#include "./inc/motif.h"

#include "./utils/cuda_utils.cuh"

int main(int argc, const char * argv[]) {
    if (argc != 6) {
        cout << "Server wrong input parameters!" << endl;
        exit(1);
    }

    string filepath = argv[1]; // Path to the graph file. The graph should be represented as an adjacency list with space separators
    string motifPath = argv[2]; //Path to motif file. The motif should be represented as edge list with space sperators.
    ui k = atoi(argv[3]);
    ui pSize = atoi(argv[4]);
    ui cpSize = atoi(argv[5]);

    Graph graph = Graph(filepath);

    //Motif M = Motif(motifPath);

    vector<ui> listingOrder;
    listingOrder.resize(graph.n);
    graph.getListingOrder(listingOrder);

    memoryAllocationGraph(deviceGraph, graph);
    memoryAllocationDAG(deviceDAG, graph.n, graph.m);

    // THIS PART IS RELATED TO GENERATING DAG

    ui *listOrder;
    chkerr(hipMalloc((void**)&(listOrder), graph.n * sizeof(ui)));
    chkerr(hipMemcpy(listOrder, listingOrder.data(), graph.n * sizeof(ui), hipMemcpyHostToDevice));

    //Tested

    
    // Get out degree in DAG
    generateDegreeDAG<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *deviceDAG, listOrder, G.n, G.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree of DAG");

    //copy out degree to offset
    chkerr(hipMemset(deviceDAG->neighbors, 0, sizeof(ui)));
    chkerr(hipMemcpy(deviceDAG->offset + 1, deviceDAG->degree, (G.n) * sizeof(ui), hipMemcpyDeviceToDevice));

    // cummulative sum offset
    thrust::inclusive_scan(thrust::device_ptr<ui>(deviceDAG->offset), thrust::device_ptr<ui>(deviceDAG->offset + G.n + 1), thrust::device_ptr<ui>(deviceDAG->offset));

    // Write neighbors of DAG
    generateNeighborDAG<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *deviceDAG, listOrder, G.n, G.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbors of DAG");
    chkerr(hipFree(listOrder));


    // THIS PART IS ABOUT CLIQUE LISTING ALGORITHM

    int maxDegree = 0;
    ui maxBitMask = memoryAllocationlevelData(*levelData, k, pSize, cpSize, maxDegree, TOTAL_WARPS);
    int level = 0;
    int iterK = k;

    ui *labels;
    chkerr(hipMalloc((void**)&(labels), G.n * sizeof(ui)));
    thrust::device_ptr<ui> dev_labels(labels);
    thrust::fill(dev_labels, dev_labels + G.n, iterK);

    chkerr(hipMemcpy(deviceGraph->degree, G.degree.data(), G.n * sizeof(ui), hipMemcpyHostToDevice));

    //TODO SHARED MEMORY 
    listIntialCliques<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, labels, iterK, G.n, G.m, pSize, cpSize, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Generate Intial Partial Cliques");

    iterK--;
    level++;
    ui offsetPartitionSize = ((pSize / (k-1)) + 1);

    createLevelDataOffset(*levelData, offsetPartitionSize, TOTAL_WARPS);
    
    flushParitions<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, pSize, cpSize, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Flush Partition data structure");

    int totalTasks;
    chkerr(hipMemcpy(&totalTasks, levelData->count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));

    while(iterK > 2) {
        thrust::fill(dev_labels, dev_labels + G.n, iterK);
        listMidCliques<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, labels, k, iterK, G.n, G.m, pSize, cpSize, maxBitMask, totalTasks, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Generate Mid Partial Cliques");

        createLevelDataOffset(*levelData, offsetPartitionSize, TOTAL_WARPS);

        flushParitions<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, pSize, cpSize, k, maxBitMask, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Flush Partition data structure");

        chkerr(hipMemcpy(&totalTasks, levelData->count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));
        iterK--;
        level++;
    }

    ui t;
    //TODO: decide the total number cliques
    chkerr(hipFree(labels));
    freeLevelPartitionData(*levelData);
    
    memoryAllocationTrie(*cliqueData, t, k);
    int totalCliques;

    chkerr(hipMalloc((void**)&totalCliques, sizeof(ui)));
    chkerr(hipMemset(totalCliques, 0, sizeof(ui)));

    if(iterK == 2) {
        writeFinalCliques<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *deviceDAG, *levelData, *cliqueData, totalCliques, k, iterK, G.n, G.m, pSize, cpSize, maxBitMask, totalTasks, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Generate Full Cliques");
    }

    freeLevelData(*levelData);

    sortTrieData<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *cliqueData, totalCliques, k, TOTAL_THREAD);
    CUDA_CHECK_ERROR("Sort Trie Data Structure");

    freeDAG(*deviceDAG);

    // TODO: reorder Trie by motif degree

    //TODO:  CLIQUE CORE DECOMPOSE

    //TODO: LOCATE CORE

    //TODO: LISTING AGAIN

    //TODO: EDGE PRUNING

    //TODO: COMPONENT DECOMPOSE

    //TODO: DYNAMIC CORE EXACT

    freeGraph(*deviceGraph);
    delete deviceGraph;
    delete deviceDAG;
    delete levelData;
    delete cliqueData;
    return 0;
}