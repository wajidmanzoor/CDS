#include "./inc/common.h"
#include "./inc/graph.h"
#include "./inc/motif.h"



#define CUDA_CHECK_ERROR(kernelName) { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA Error in kernel %s, file %s at line %d: %s\n", kernelName, __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

bool fileExists(const string& filename) {
    struct stat buffer;
    return (stat(filename.c_str(), &buffer) == 0);
}

void createLevelDataOffset(cliqueLevelDataPointer levelData, ui offsetPartitionSize, ui TOTAL_WARPS) {
    thrust::transform(thrust::device, thrust::make_counting_iterator(0), thrust::make_counting_iterator(TOTAL_WARPS), levelData.temp + 1,
                      [=] __device__ (int i) {
                          int task_count = levelData.count[i];
                          return (task_count > 0) ? levelData.offsetPartition[i * offsetPartitionSize + task_count] : 0;
                      });

    thrust::inclusive_scan(thrust::device, levelData.temp, levelData.temp + TOTAL_WARPS + 1, levelData.temp);
    thrust::inclusive_scan(thrust::device, levelData.count, levelData.count + TOTAL_WARPS + 1, levelData.count);
}

void writeOrAppend(const string& filename, const string& data) {
    ofstream file;
    
    // Check if the file exists
    if (fileExists(filename)) {
        // Open the file in append mode if it exists
        file.open(filename, ios::app);
    } else {
        // Open the file in write mode if it doesn't exist
        file.open(filename);
    }
    
    if (file.is_open()) {
        file << data << endl;
        file.close();
    } else {
        cerr << "Unable to open the file." << endl;
    }
}

int main(int argc, const char * argv[]) {
    if (argc != 6) {
        cout << "Server wrong input parameters!" << endl;
        exit(1);
    }

    string filepath = argv[1]; // Path to the graph file. The graph should be represented as an adjacency list with space separators
    string motifPath = argv[2]; //Path to motif file. The motif should be represented as edge list with space sperators.
    ui k = atoi(argv[3]);
    ui pSize = atoi(argv[4]);
    ui cpSize = atoi(argv[5]);

    Graph graph = Graph(filepath);

    //Motif M = Motif(motifPath);

    vector<ui> listingOrder;
    listingOrder.resize(graph.n);
    graph.getListingOrder(listingOrder);

    //Tested
    
    G.getListingOrder(listingOrder);
    memoryAllocationGraph(*deviceGraph, G);
    memoryAllocationDAG(*deviceDAG, G.n, G.m);

    // THIS PART IS RELATED TO GENERATING DAG

    ui *listOrder;
    chkerr(hipMalloc((void**)&(listOrder), G.n * sizeof(ui)));
    chkerr(hipMemcpy(listOrder, listingOrder.data(), G.n * sizeof(ui), hipMemcpyHostToDevice));

    // Get out degree in DAG
    generateDegreeDAG<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *deviceDAG, listOrder, G.n, G.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree of DAG");

    //copy out degree to offset
    chkerr(hipMemset(deviceDAG->neighbors, 0, sizeof(ui)));
    chkerr(hipMemcpy(deviceDAG->offset + 1, deviceDAG->degree, (G.n) * sizeof(ui), hipMemcpyDeviceToDevice));

    // cummulative sum offset
    thrust::inclusive_scan(thrust::device_ptr<ui>(deviceDAG->offset), thrust::device_ptr<ui>(deviceDAG->offset + G.n + 1), thrust::device_ptr<ui>(deviceDAG->offset));

    // Write neighbors of DAG
    generateNeighborDAG<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *deviceDAG, listOrder, G.n, G.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbors of DAG");
    chkerr(hipFree(listOrder));


    // THIS PART IS ABOUT CLIQUE LISTING ALGORITHM

    int maxDegree = 0;
    ui maxBitMask = memoryAllocationlevelData(*levelData, k, pSize, cpSize, maxDegree, TOTAL_WARPS);
    int level = 0;
    int iterK = k;

    ui *labels;
    chkerr(hipMalloc((void**)&(labels), G.n * sizeof(ui)));
    thrust::device_ptr<ui> dev_labels(labels);
    thrust::fill(dev_labels, dev_labels + G.n, iterK);

    chkerr(hipMemcpy(deviceGraph->degree, G.degree.data(), G.n * sizeof(ui), hipMemcpyHostToDevice));

    //TODO SHARED MEMORY 
    listIntialCliques<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, labels, iterK, G.n, G.m, pSize, cpSize, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Generate Intial Partial Cliques");

    iterK--;
    level++;
    ui offsetPartitionSize = ((pSize / (k-1)) + 1);

    createLevelDataOffset(*levelData, offsetPartitionSize, TOTAL_WARPS);
    
    flushParitions<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, pSize, cpSize, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Flush Partition data structure");

    int totalTasks;
    chkerr(hipMemcpy(&totalTasks, levelData->count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));

    while(iterK > 2) {
        thrust::fill(dev_labels, dev_labels + G.n, iterK);
        listMidCliques<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, labels, k, iterK, G.n, G.m, pSize, cpSize, maxBitMask, totalTasks, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Generate Mid Partial Cliques");

        createLevelDataOffset(*levelData, offsetPartitionSize, TOTAL_WARPS);

        flushParitions<<<BLK_NUMS, BLK_DIM>>>(*deviceDAG, *levelData, pSize, cpSize, k, maxBitMask, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Flush Partition data structure");

        chkerr(hipMemcpy(&totalTasks, levelData->count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));
        iterK--;
        level++;
    }

    ui t;
    //TODO: decide the total number cliques
    chkerr(hipFree(labels));
    freeLevelPartitionData(*levelData);
    
    memoryAllocationTrie(*cliqueData, t, k);
    int totalCliques;

    chkerr(hipMalloc((void**)&totalCliques, sizeof(ui)));
    chkerr(hipMemset(totalCliques, 0, sizeof(ui)));

    if(iterK == 2) {
        writeFinalCliques<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *deviceDAG, *levelData, *cliqueData, totalCliques, k, iterK, G.n, G.m, pSize, cpSize, maxBitMask, totalTasks, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Generate Full Cliques");
    }

    freeLevelData(*levelData);

    sortTrieData<<<BLK_NUMS, BLK_DIM>>>(*deviceGraph, *cliqueData, totalCliques, k, TOTAL_THREAD);
    CUDA_CHECK_ERROR("Sort Trie Data Structure");

    freeDAG(*deviceDAG);

    // TODO: reorder Trie by motif degree

    //TODO:  CLIQUE CORE DECOMPOSE

    //TODO: LOCATE CORE

    //TODO: LISTING AGAIN

    //TODO: EDGE PRUNING

    //TODO: COMPONENT DECOMPOSE

    //TODO: DYNAMIC CORE EXACT

    freeGraph(*deviceGraph);
    delete deviceGraph;
    delete deviceDAG;
    delete levelData;
    delete cliqueData;
    return 0;
}