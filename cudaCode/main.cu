#include "hip/hip_runtime.h"
#include "./inc/common.h"
#include "./inc/graph.h"

#include "./utils/cuda_utils.cuh"
#include "./inc/gpuMemoryAllocation.cuh"
#include "./inc/helpers.cuh"
#include <thrust/count.h>

#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/binary_search.h>
#include <thrust/async/copy.h>
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>


bool DEBUG = true;

__global__ void printmap(densestCorePointer densestCore, ui coreSize){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<coreSize){
    printf("idx %d map %d \n",idx,densestCore.mapping[idx]);
  }
}

void generateDAG(const Graph& graph,deviceGraphPointers& deviceGraph,deviceDAGpointer& deviceDAG , vector<ui> listingOrder){
    memoryAllocationDAG(deviceDAG, graph.n, graph.m);


    ui *listOrder;
    chkerr(hipMalloc((void**)&(listOrder), graph.n * sizeof(ui)));
    chkerr(hipMemcpy(listOrder, listingOrder.data(), graph.n * sizeof(ui), hipMemcpyHostToDevice));

    // Get out degree in DAG
    generateDegreeDAG<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, deviceDAG, listOrder, graph.n, graph.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree of DAG");

    //copy out degree to offset
    chkerr(hipMemset(deviceDAG.offset, 0, sizeof(ui)));
    chkerr(hipMemcpy(deviceDAG.offset + 1, deviceDAG.degree, (graph.n) * sizeof(ui), hipMemcpyDeviceToDevice));

    // cummulative sum offset
    thrust::inclusive_scan(thrust::device_ptr<ui>(deviceDAG.offset), thrust::device_ptr<ui>(deviceDAG.offset + graph.n + 1), thrust::device_ptr<ui>(deviceDAG.offset));


    if(DEBUG){
        ui *h_degree,*h_offset;
        h_degree = new ui[graph.n];
        h_offset = new ui[graph.n + 1];

        chkerr(hipMemcpy(h_degree, deviceDAG.degree, graph.n * sizeof(ui), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(h_offset, deviceDAG.offset, (graph.n + 1) * sizeof(ui), hipMemcpyDeviceToHost));

        cout<<endl<<endl<<"DAG DATA"<<endl;
        cout<<endl<<"DAG"<<endl<<"Degree ";
        for(int i = 0; i < graph.n; i++) {
            cout << h_degree[i] << " ";
        }
        cout<<endl<<"offset ";
        for(int i = 0; i < graph.n + 1; i++) {
            cout << h_offset[i] << " ";
        }
        cout<<endl;

    }
    

    // Write neighbors of DAG
    size_t sharedMemoryGenDagNeig =  WARPS_EACH_BLK * sizeof(ui);
    generateNeighborDAG<<<BLK_NUMS, BLK_DIM,sharedMemoryGenDagNeig>>>(deviceGraph, deviceDAG, listOrder, graph.n, graph.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbor of DAG");

    if(DEBUG){
        ui *h_neighbors;
        h_neighbors = new ui[graph.m];
        chkerr(hipMemcpy(h_neighbors, deviceDAG.neighbors, graph.m * sizeof(ui), hipMemcpyDeviceToHost));
        cout<<"neigh ";
        for(int i = 0; i < graph.m; i++) {
            cout << h_neighbors[i] << " ";
        }
        cout<<endl;


    }
    


    chkerr(hipFree(listOrder));


}

ui listAllCliques(const Graph& graph,deviceGraphPointers& deviceGraph,deviceDAGpointer& deviceDAG,cliqueLevelDataPointer levelData, ui k, ui pSize,ui  cpSize, ui t){
    thrust::device_ptr<ui> dev_degree(deviceDAG.degree);
    auto max_iter = thrust::max_element(dev_degree, dev_degree + graph.n);
    int maxDegree = *max_iter;
    ui maxBitMask = memoryAllocationlevelData(levelData, k, pSize, cpSize, maxDegree, TOTAL_WARPS);

    int level = 0;
    int iterK = k;

    ui *labels;
    chkerr(hipMalloc((void**)&(labels), (graph.n * TOTAL_WARPS) * sizeof(ui)));
    thrust::device_ptr<ui> dev_labels(labels);
    thrust::fill(dev_labels, dev_labels + graph.n*TOTAL_WARPS, iterK);

    chkerr(hipMemcpy(deviceGraph.degree, graph.degree.data(), graph.n * sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemset(levelData.partialCliquesPartition, 0,  (TOTAL_WARPS * pSize)* sizeof(ui)));

    size_t sharedMemoryIntialClique =  WARPS_EACH_BLK * sizeof(ui);
    listIntialCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryIntialClique>>>(deviceDAG, levelData, labels, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Generate Intial Partial Cliques");

    ui partialSize = TOTAL_WARPS * pSize;
    //ui candidateSize = TOTAL_WARPS * cpSize;
    ui offsetSize = ((pSize / (k - 1)) + 1) * TOTAL_WARPS;


    ui offsetPartitionSize = ((pSize / (k-1)) + 1);

    createLevelDataOffset(levelData, offsetPartitionSize, TOTAL_WARPS);
    flushParitions<<<BLK_NUMS, BLK_DIM>>>(deviceDAG, levelData, pSize,cpSize,k, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Flush Partition data structure");

    iterK--;
    level++;

    int totalTasks;
    chkerr(hipMemcpy(&totalTasks, levelData.count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));
    size_t sharedMemoryMid =  WARPS_EACH_BLK * sizeof(ui);
    while(iterK > 2) {
      thrust::device_ptr<ui> dev_labels(labels);
      thrust::fill(dev_labels, dev_labels + graph.n*TOTAL_WARPS, iterK);
      chkerr(hipMemset(levelData.count, 0, (TOTAL_WARPS + 1) * sizeof(ui)));
      chkerr(hipMemset(levelData.temp, 0, (TOTAL_WARPS + 1) * sizeof(ui)));
      chkerr(hipMemset(levelData.offsetPartition, 0,  (offsetSize)* sizeof(ui)));
      chkerr(hipMemset(levelData.validNeighMaskPartition,0, (partialSize * maxBitMask) * sizeof(ui)));
      listMidCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryMid>>>(deviceDAG, levelData, labels, k, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, totalTasks, level, TOTAL_WARPS);
      CUDA_CHECK_ERROR("Generate Mid Partial Cliques");


      createLevelDataOffset(levelData, offsetPartitionSize, TOTAL_WARPS);

      chkerr(hipMemset(levelData.offset,0,offsetSize*sizeof(ui)));
      chkerr(hipMemset(levelData.validNeighMask,0,partialSize*maxBitMask*sizeof(ui)));
      flushParitions<<<BLK_NUMS, BLK_DIM>>>(deviceDAG, levelData, pSize,cpSize,k, maxBitMask, level, TOTAL_WARPS);
      CUDA_CHECK_ERROR("Flush Partition data structure");

      
      chkerr(hipMemcpy(&totalTasks, levelData.count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));

      iterK--;
      level++;
    }


    chkerr(hipFree(labels));
    memoryAllocationTrie(cliqueData, t, k);
    ui *totalCliques;

    chkerr(hipMalloc((void**)&totalCliques, sizeof(ui)));
    chkerr(hipMemset(totalCliques, 0, sizeof(ui)));
    size_t sharedMemoryFinal =  WARPS_EACH_BLK * sizeof(ui);


    thrust::device_ptr<int> dev_ptr(cliqueData.status);
    thrust::fill(dev_ptr, dev_ptr + t, -2);

    chkerr(hipMemset(cliqueData.trie, 0, t * k * sizeof(ui)));
    if(iterK == 2) {
        writeFinalCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryFinal>>>(deviceGraph, deviceDAG, levelData, cliqueData, totalCliques, k, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, t,totalTasks, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Generate Full Cliques");
    }

    if(DEBUG){
        int *h_cliques,*status;
        h_cliques = new int[t*k];
        status = new int[t];
        chkerr(hipMemcpy(h_cliques, cliqueData.trie, k * t * sizeof(ui), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(status, cliqueData.status, t * sizeof(ui), hipMemcpyDeviceToHost));
        cout<<endl;

        cout<<endl<<"Cliques Data "<<endl;
        for(int i =0;i<k;i++){
        cout<<endl<<"CL "<<i<<"  ";
        for(int j =0;j<t;j++){
            cout<<h_cliques[i*t+j]<<" ";
        }
        }
        cout<<endl<<"stat  ";
        for(int i = 0; i < t; i++) {
            cout << status[i] << " ";

        }

        ui *h_cdegree;
        h_cdegree = new ui[graph.n];
        //ui v = 4;

        //chkerr(hipMemcpy(deviceGraph.cliqueDegree + 7,&v ,  sizeof(ui), hipMemcpyHostToDevice));
        chkerr(hipMemcpy(h_cdegree, deviceGraph.cliqueDegree, graph.n* sizeof(ui), hipMemcpyDeviceToHost));


        cout<<endl<<"Clique Degree"<<endl;

        for(int i = 0; i < graph.n; i++) {
            cout <<i<<" ";
        }
        cout<<endl;
        for(int i = 0; i < graph.n; i++) {
            cout << h_cdegree[i] << " ";
        }

        

        //size_t sharedMemorySort =  2*k*WARPS_EACH_BLK * sizeof(ui);
        //sortTrieData<<<BLK_NUMS, BLK_DIM,sharedMemorySort>>>(deviceGraph, cliqueData, tt,t, k, TOTAL_THREAD);
        //CUDA_CHECK_ERROR("Sort Trie Data Structure");


        cout<<endl;

    }
    ui tt;
    chkerr(hipMemcpy(&tt, totalCliques, sizeof(ui), hipMemcpyDeviceToHost));
    
    freeLevelData(levelData);
    freeLevelPartitionData(levelData);
    freeDAG(deviceDAG);

    return tt;

}

void cliqueCoreDecompose(const Graph& graph,deviceGraphPointers& deviceGraph,deviceCliquesPointer& cliqueData, ui &maxCore, double &maxDensity, ui &coreSize, ui &coreTotalCliques, ui glBufferSize, ui k, ui t, ui tt){
    ui level = 0;
    ui count = 0;
    ui *globalCount = NULL;
    ui *bufTails  = NULL;
    ui *glBuffers = NULL;

    chkerr(hipMalloc((void**)&(globalCount), sizeof(ui)));
    chkerr(hipMalloc((void**)&(bufTails), BLK_NUMS*sizeof(ui)));
    chkerr(hipMalloc((void**)&(glBuffers), BLK_NUMS*glBufferSize*sizeof(ui)));
    chkerr(hipMemset(globalCount, 0, sizeof(ui)));
    hipDeviceSynchronize();



    chkerr(hipMemcpy(deviceGraph.cliqueCore, deviceGraph.cliqueDegree, graph.n * sizeof(ui), hipMemcpyDeviceToDevice));


    thrust::device_vector<int> dev_vec(cliqueData.status, cliqueData.status + t);

    ui currentCliques = thrust::count(dev_vec.begin(), dev_vec.end(), -1);
    double currentDensity = static_cast<double>(currentCliques) / (graph.n - count);

    maxDensity = currentDensity;
    maxCore = 0;
    coreTotalCliques = currentCliques;
    coreSize = graph.n;


    while(count < graph.n){
        hipMemset(bufTails, 0, sizeof(ui)*BLK_NUMS);

        // Select nodes whoes current degree is level, that means they should be removed as part of the level core
        selectNodes<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, bufTails, glBuffers, glBufferSize, graph.n, level);
        hipDeviceSynchronize();

        //Total number of verticies in buffer
        thrust::device_vector<ui> dev_vec1(bufTails, bufTails + BLK_NUMS);
        ui sum = thrust::reduce(dev_vec1.begin(), dev_vec1.end(), 0, thrust::plus<ui>());
        hipDeviceSynchronize();


        processNodesByWarp<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, cliqueData , bufTails, glBuffers, globalCount, glBufferSize, graph.n, level, k, t, tt);
        hipDeviceSynchronize();

        chkerr(hipMemcpy(&count, globalCount, sizeof(unsigned int), hipMemcpyDeviceToHost));

        level++;
        thrust::device_vector<int> dev_vec2(cliqueData.status, cliqueData.status + t);


       if((graph.n - count)!=0){
        currentCliques = thrust::count(dev_vec2.begin(), dev_vec2.end(), -1);
        currentDensity = static_cast<double>(currentCliques) / (graph.n - count);

        if(currentDensity>=maxDensity){
            maxDensity = currentDensity;
            maxCore = level;
            coreTotalCliques = currentCliques;
            coreSize = graph.n-count;

        }

       }

        hipDeviceSynchronize();

    }
    hipFree(globalCount);
    hipFree(bufTails);
    hipFree(glBuffers);

}

ui generateDensestCore(const Graph& graph,deviceGraphPointers& deviceGraph, densestCorePointer &densestCore, ui coreSize, ui coreTotalCliques, ui lowerBoundDensity){
    memoryAllocationDensestCore(densestCore, coreSize, lowerBoundDensity , coreTotalCliques);

    ui *globalCount;

    chkerr(hipMalloc((void**)&globalCount, sizeof(ui)));
    chkerr(hipMemset(globalCount, 0, sizeof(ui)));

    generateDensestCore<<<BLK_NUMS, BLK_DIM>>>(deviceGraph,densestCore,globalCount,graph.n,lowerBoundDensity,TOTAL_WARPS);
    hipDeviceSynchronize();

    ui *h_offset;
    h_offset = new ui[coreSize+1];

    chkerr(hipMemcpy(h_offset, densestCore.offset, (coreSize+1) * sizeof(ui), hipMemcpyDeviceToHost));

     cout<<endl<<"offset b ";
    for(int i=0;i<=coreSize;i++){
      cout<<h_offset[i]<<" ";
    }
    cout<<endl;



    thrust::inclusive_scan(thrust::device_ptr<ui>(densestCore.offset), thrust::device_ptr<ui>(densestCore.offset + coreSize + 1), thrust::device_ptr<ui>(densestCore.offset));

    //debug
    ui *h_mapping;
    h_mapping = new ui[coreSize];
    chkerr(hipMemcpy(h_mapping, densestCore.mapping, coreSize * sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(h_offset, densestCore.offset, (coreSize+1) * sizeof(ui), hipMemcpyDeviceToHost));


    cout<<endl<<"Densest core data "<<endl;
    cout<<endl<<"mapping ";
    for(int i=0;i<coreSize;i++){
      cout<<h_mapping[i]<<" ";
    }
    cout<<endl<<"offset ";
    for(int i=0;i<=coreSize;i++){
      cout<<h_offset[i]<<" ";
    }
    cout<<endl;


    ui edgeCountCore;
    chkerr(hipMemcpy(&edgeCountCore, densestCore.offset+coreSize , sizeof(ui), hipMemcpyDeviceToHost));

    cout<<"edgeCountCore "<<edgeCountCore<<endl;

    chkerr(hipMemcpy(densestCore.m,&edgeCountCore, sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMalloc((void**)&(densestCore.neighbors), edgeCountCore * sizeof(ui)));

    thrust::device_ptr<unsigned int> d_vertex_map_ptr(densestCore.mapping);
    thrust::device_ptr<unsigned int> d_reverse_map_ptr(densestCore.reverseMap);


    thrust::device_vector<unsigned int> d_indices(coreSize);
    thrust::sequence(d_indices.begin(), d_indices.end());
    // Scatter indices into the reverse mapping array
    thrust::scatter(d_indices.begin(), d_indices.end(), d_vertex_map_ptr, d_reverse_map_ptr);


    cout<<"gaph size "<<graph.n<<" core size "<<coreSize<<endl;


    size_t sharedMemoryGenNeighCore =  WARPS_EACH_BLK * sizeof(ui);
    generateNeighborDensestCore<<<BLK_NUMS, BLK_DIM,sharedMemoryGenNeighCore>>>(deviceGraph,densestCore,lowerBoundDensity,TOTAL_WARPS);
    hipDeviceSynchronize();

    //Debug
    ui *h_neighbors;
    h_neighbors = new ui[edgeCountCore];
    chkerr(hipMemcpy(h_neighbors, densestCore.neighbors, edgeCountCore * sizeof(ui), hipMemcpyDeviceToHost));

    cout<<endl<<"neighbors ";
    for(int i=0;i<edgeCountCore;i++){
      cout<<h_neighbors[i]<<" ";
    }
    cout<<endl;


    return edgeCountCore;


}
ui prune(densestCorePointer &densestCore, deviceCliquesPointer &cliqueData, devicePrunedNeighbors &prunedNeighbors,
    ui vertexCount, ui edgecount, ui k, ui t, ui tt, ui lowerBoundDensity) {

    // Allocate and initialize pruneStatus
    thrust::device_ptr<ui> d_pruneStatus(prunedNeighbors.pruneStatus);
    thrust::fill(d_pruneStatus, d_pruneStatus + edgecount, 1);

    // Kernel to determine pruning

    pruneEdges<<<BLK_NUMS, BLK_DIM>>>(densestCore, cliqueData,prunedNeighbors.pruneStatus, t, tt, k, lowerBoundDensity);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Get prune status of each edge");

    if(DEBUG){
        ui *h_pstatus = new ui[edgecount];
        chkerr(hipMemcpy(h_pstatus, prunedNeighbors.pruneStatus, edgecount * sizeof(ui), hipMemcpyDeviceToHost));

        cout<<endl<<"Neigh Status"<<endl;
        for (ui i = 0; i < edgecount; i++) {
            std::cout << h_pstatus[i] << " ";
        }
        std::cout << std::endl;

        delete[] h_pstatus;

    }
    


    // Allocate and initialize newOffset

    // Kernel to generate out-degrees
    generateDegreeAfterPrune<<<BLK_NUMS, BLK_DIM>>>(
        densestCore, prunedNeighbors.pruneStatus, prunedNeighbors.newOffset, vertexCount, edgecount, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree after pruning");

    // Inclusive scan to build offset array
    thrust::inclusive_scan(
        thrust::device_pointer_cast(prunedNeighbors.newOffset),
        thrust::device_pointer_cast(prunedNeighbors.newOffset + vertexCount + 1),
        thrust::device_pointer_cast(prunedNeighbors.newOffset));

    // Get total number of remaining edges
    ui newEdgeCount;
    chkerr(hipMemcpy(&newEdgeCount, prunedNeighbors.newOffset + vertexCount, sizeof(ui), hipMemcpyDeviceToHost));

    // Allocate memory for newNeighbors
    chkerr(hipMalloc((void**)&(prunedNeighbors.newNeighbors), newEdgeCount * sizeof(ui)));

    // Kernel to generate neighbors list
    size_t sharedMemoryGenNeig = WARPS_EACH_BLK * sizeof(ui);
    generateNeighborAfterPrune<<<BLK_NUMS, BLK_DIM, sharedMemoryGenNeig>>>(
        densestCore, prunedNeighbors.pruneStatus, prunedNeighbors.newOffset, prunedNeighbors.newNeighbors, vertexCount, edgecount, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbor after prune");


    if(DEBUG){
        ui *h_offset, *h_neigh;
        h_offset = new ui[vertexCount+1];
        h_neigh = new ui[newEdgeCount];
        chkerr(hipMemcpy(h_offset, prunedNeighbors.newOffset, (vertexCount+1) * sizeof(ui), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(h_neigh, prunedNeighbors.newNeighbors, newEdgeCount * sizeof(ui), hipMemcpyDeviceToHost));

        cout<<endl<<"Data After Pruning "<<endl;
        cout<<"in offset";
        for(ui i=0;i<vertexCount+1;i++){
        cout<<h_offset[i]<<" ";
        }cout<<endl;
        cout<<"in neigh";
        for(ui i=0;i<newEdgeCount;i++){
        cout<<h_neigh[i]<<" ";
        }cout<<endl;

    }
    

    return newEdgeCount;
}

int componentDecompose(deviceComponentPointers &conComp,devicePrunedNeighbors &prunedNeighbors, ui vertexCount, ui edgecount){

    ui *changed;
    chkerr(hipMalloc((void**)&changed, sizeof(ui)));
    chkerr(hipMemset(changed, 0 , sizeof(ui)));

    thrust::device_ptr<ui> components = thrust::device_pointer_cast(conComp.components);
    thrust::sequence(components, components + vertexCount);

    //int iter = 0;
    //can be used to put a limit on num iters
    ui hostChanged;
    do{
      chkerr(hipMemset(changed, 0 , sizeof(ui)));
      componentDecomposek<<<BLK_NUMS, BLK_DIM>>>(conComp, prunedNeighbors, changed, vertexCount, edgecount, TOTAL_WARPS);
      hipDeviceSynchronize();
      CUDA_CHECK_ERROR("Coponenet Decompose");

      chkerr(hipMemcpy(&hostChanged,changed , sizeof(ui), hipMemcpyDeviceToHost));

    }while(hostChanged>0);

    // unique component
    thrust::device_vector<int> uniqueComponents(vertexCount);
    auto new_end = thrust::unique_copy(components , components + vertexCount,
                                   uniqueComponents.begin());
    int totalComponents = new_end - uniqueComponents.begin();

    //Create component offset
    thrust::device_ptr<ui> componentOffsets(conComp.componentOffset);
    thrust::lower_bound(components , components + vertexCount,
                    uniqueComponents.begin(), uniqueComponents.begin() + totalComponents,
                    componentOffsets);
    componentOffsets[totalComponents] = vertexCount;

    uniqueComponents.resize(totalComponents);


    thrust::sort(uniqueComponents.begin(), uniqueComponents.end());


    thrust::lower_bound(
        uniqueComponents.begin(), uniqueComponents.end(),
        components, components + vertexCount,
        components // In-place remap
    );

    //Vertices in Densest Core, use mapping to get actual verticies
    thrust::device_ptr<ui> vertices(densestCore.mapping);

    thrust::sequence(thrust::device_pointer_cast(conComp.mapping), thrust::device_pointer_cast(conComp.mapping + vertexCount));

    thrust::sort_by_key(
      components,
      components + vertexCount,
      thrust::device_pointer_cast(conComp.mapping)
    );
    //TODO : new neighbor list, new neighbor offset

     return totalComponents;

}









int main(int argc, const char * argv[]) {
    if (argc != 8) {
        cout << "Server wrong input parameters!" << endl;
        exit(1);
    }

    string filepath = argv[1]; // Path to the graph file. The graph should be represented as an adjacency list with space separators
    string motifPath = argv[2]; //Path to motif file. The motif should be represented as edge list with space sperators.
    ui k = atoi(argv[3]);
    ui pSize = atoi(argv[4]);
    ui cpSize = atoi(argv[5]);
    ui glBufferSize = atoi(argv[6]);
    ui partitionSize = atoi(argv[7]);

    if(DEBUG){
        cout << "filepath: " << filepath << endl;
        cout << "motifPath: " << motifPath << endl;
        cout <<"k: " << k << endl;
        cout << "pSize: " << pSize << endl;
        cout << "cpSize: " << cpSize << endl;
    }
    //find a way to do this
    ui t = 10;
    
    Graph graph = Graph(filepath);

    //Print Graph
    if(DEBUG){
        cout<<"Graph Data "<<endl;
        cout<<"Graph"<<endl<<"Offset: ";
        for (int i = 0; i < (graph.n +1); i++) {
            cout<<graph.offset[i]<<" ";

        }
        cout<<endl<<"Neighbors: ";
        for (int i = 0; i < 2*graph.m; i++) {
            cout<<graph.neighbors[i]<<" ";
        }
        cout<<endl;
        cout<<"Degree: ";
        for (int i = 0; i < graph.n; i++) {
            cout<<graph.degree[i]<<" ";
        }
        cout<<endl;

    }
    
    vector<ui> listingOrder;
    listingOrder.resize(graph.n);
    graph.getListingOrder(listingOrder);

    if(DEBUG){
        cout<<endl<<endl<<"Listing Order: ";
        for (int i = 0; i < graph.n; i++) {
            cout << listingOrder[i] << " ";
        }
        cout<<endl<<"Core ";
        for(int i = 0; i < graph.n; i++) {
            cout << graph.core[i] << " ";
        }
        cout<<endl<<"Peel Seq ";
        for(int i = 0; i < graph.n; i++) {
            cout  << graph.corePeelSequence[i] << " ";
        }
        cout<<endl;

    }

    memoryAllocationGraph(deviceGraph, graph);

    generateDAG(graph,deviceGraph, deviceDAG,listingOrder);

    ui tt = listAllCliques(graph, deviceGraph, deviceDAG, levelData, k, pSize, cpSize,t);

    ui coreSize, coreTotalCliques,maxCore;
    double maxDensity;
    cliqueCoreDecompose(graph,deviceGraph,cliqueData,maxCore, maxDensity, coreSize, coreTotalCliques,glBufferSize, k,  t, tt);

    if(DEBUG){
        cout<<endl<<"Clique data after core decompose "<<endl;
        int *h_cliques,*status;
        h_cliques = new int[t*k];
        status = new int[t];
        chkerr(hipMemcpy(h_cliques, cliqueData.trie, k * t * sizeof(ui), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(status, cliqueData.status, t * sizeof(ui), hipMemcpyDeviceToHost));
        for(int i =0;i<k;i++){
        cout<<endl<<"CL "<<i<<"  ";
        for(int j =0;j<t;j++){
            cout<<h_cliques[i*t+j]<<" ";
        }
        }
        cout<<endl<<"stat  ";
        for(int i = 0; i < t; i++) {
            cout << status[i] << " ";

        }
        cout<<endl;
    }
    
    ui lowerBoundDensity = maxCore;
 
    ui edgecount = generateDensestCore(graph,deviceGraph,  densestCore, coreSize, coreTotalCliques,lowerBoundDensity);

    ui vertexCount;
    chkerr(hipMemcpy(&vertexCount, densestCore.n, sizeof(ui), hipMemcpyDeviceToHost));

    memoryAllocationPrunnedNeighbors(prunedNeighbors, vertexCount , edgecount);

    ui newEdgeCount = prune(densestCore, cliqueData, prunedNeighbors, vertexCount, edgecount, k, t, tt, lowerBoundDensity);

    memoryAllocationComponent(conComp, vertexCount , newEdgeCount);

    ui totalComponents = componentDecompose(conComp, prunedNeighbors, vertexCount, newEdgeCount);

    //Dynamic exact

    thrust::device_ptr<unsigned int> d_vertex_map_ptr(conComp.mapping);
    thrust::device_ptr<unsigned int> d_reverse_map_ptr(conComp.reverseMapping);

    thrust::device_vector<unsigned int> d_indices(vertexCount);
    thrust::sequence(d_indices.begin(), d_indices.end());
    thrust::scatter(d_indices.begin(), d_indices.end(), d_vertex_map_ptr, d_reverse_map_ptr);


    //Counter to store total cliques of each component
    ui *compCounter;
    chkerr(hipMalloc((void**)&(compCounter), (totalComponents+1)* sizeof(ui)));
    chkerr(hipMemset(compCounter, 0, (totalComponents+1) * sizeof(ui)));

    // Get total cliques of each connected components

    getConnectedComponentStatus<<<BLK_NUMS, BLK_DIM>>>(conComp,cliqueData, densestCore,compCounter,t, tt,k, maxCore,TOTAL_THREAD);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Calculate total cliques for each Component");

    thrust::inclusive_scan(
        thrust::device_pointer_cast(compCounter),
        thrust::device_pointer_cast(compCounter + totalComponents + 1),
        thrust::device_pointer_cast(compCounter));


    ui totaLCliques;
    chkerr(hipMemcpy(&totaLCliques,compCounter+totalComponents,sizeof(ui),hipMemcpyDeviceToHost));

    // Allocate memory for new clique data arranged my connected component
    memoryAllocationTrie(finalCliqueData, totaLCliques, k);


    ui *counter;
    chkerr(hipMalloc((void**)&(counter), totalComponents* sizeof(ui)));
    chkerr(hipMemset(counter, 0, totalComponents * sizeof(ui)));
    rearrangeCliqueData<<<BLK_NUMS, BLK_DIM>>>(conComp, cliqueData,  finalCliqueData, densestCore, compCounter, counter, t,  tt,  k, totaLCliques,TOTAL_THREAD);

    if(DEBUG){
        int *h_cliques,*status;
        h_cliques = new int[t*k];
        status = new int[t];
        cout<<endl<<"Clique data after rearange "<<endl;
        chkerr(hipMemcpy(h_cliques, finalCliqueData.trie, k * totaLCliques * sizeof(ui), hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(status, finalCliqueData.status, totaLCliques* sizeof(ui), hipMemcpyDeviceToHost));

        for(int i =0;i<k;i++){
        cout<<endl<<"CL "<<i<<"  ";
        for(int j =0;j<totaLCliques;j++){
            cout<<h_cliques[i*totaLCliques+j]<<" ";
        }
        }
        cout<<endl<<"stat  ";
        for(int i = 0; i < totaLCliques; i++) {
            cout << status[i] << " ";

        }
        cout<<endl;

    }
    
    double* bounds;
    chkerr(hipMalloc((void**)&bounds, (totalComponents*2)* sizeof(double)));
    double* upperBound = bounds;
    double* lowerBound = bounds + totalComponents;


    chkerr(hipMalloc((void**)&(flowNetwork.offset), (1 + totalComponents) * sizeof(ui)));
    chkerr(hipMalloc((void**)&(flowNetwork.neighborOffset1), (1 + totalComponents) * sizeof(ui)));

    thrust::device_ptr<int> d_cliqueCore(deviceGraph.cliqueCore);

    // Find the maximum element using thrust::reduce
    int max_int = thrust::reduce(d_cliqueCore, d_cliqueCore + graph.n,
                               0, thrust::maximum<int>());

    // Convert to double and return
    double md =  static_cast<double>(max_int);

    getLbUbandSize<<<BLK_NUMS, BLK_DIM>>>( conComp, compCounter, lowerBound, upperBound, flowNetwork.offset, flowNetwork.neighborOffset1, totalComponents, k, md);

    thrust::inclusive_scan(
        thrust::device_pointer_cast(flowNetwork.offset),
        thrust::device_pointer_cast(flowNetwork.offset + totalComponents + 1),
        thrust::device_pointer_cast(flowNetwork.offset));
    thrust::inclusive_scan(
        thrust::device_pointer_cast(flowNetwork.neighborOffset1),
        thrust::device_pointer_cast(flowNetwork.neighborOffset1 + totalComponents + 1),
        thrust::device_pointer_cast(flowNetwork.neighborOffset1));

    ui flownetworkVertexSize, flownetworkNeighborSize;
    chkerr(hipMemcpy(&flownetworkVertexSize, flowNetwork.offset+totalComponents, sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(&flownetworkNeighborSize, flowNetwork.neighborOffset1+totalComponents, sizeof(ui), hipMemcpyDeviceToHost));

    memoryAllocationFlowNetwork(flowNetwork, flownetworkVertexSize, flownetworkNeighborSize, totalComponents);

    if(DEBUG){
        double *lb, *ub;
        lb = new double[totalComponents];
        ub = new double[totalComponents];

        ui *h_ccoffset, *h_neighborSize;

        h_ccoffset = new ui[totalComponents+1];
        h_neighborSize = new ui[totalComponents+1];

        chkerr(hipMemcpy(lb,lowerBound,totalComponents*sizeof(double),hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(ub,upperBound,totalComponents*sizeof(double),hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(h_ccoffset,flowNetwork.offset,(totalComponents+1)*sizeof(ui),hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(h_neighborSize,flowNetwork.neighborOffset1,(totalComponents+1)*sizeof(ui),hipMemcpyDeviceToHost));

        cout<<endl<<"FLOW NETWORK"<<endl;
        cout<<"Lower Bound ";
        for(ui i=0;i<totalComponents;i++){
        cout<<lb[i]<<" ";
        }
        cout<<endl;

        cout<<"Upper Bound ";
        for(ui i=0;i<totalComponents;i++){
        cout<<ub[i]<<" ";
        }
        cout<<endl;
        cout<<"Vertex Offset ";
        for(ui i=0;i<=totalComponents;i++){
        cout<<h_ccoffset[i]<<" ";
        }
        cout<<endl;
        cout<<"Neigh Size ";
        for(ui i=0;i<=totalComponents;i++){
        cout<<h_neighborSize[i]<<" ";
        }
        cout<<endl;
    }
    thrust::device_ptr<double> dev_lowerBound(lowerBound);
     // Find the maximum element in the lowerBound array
    thrust::device_ptr<double> max_iter = thrust::max_element(dev_lowerBound, dev_lowerBound + totalComponents);

    // Copy the result back to host if needed
    double max_lowerBound = *max_iter;

    //double just  = 1.0;

    createFlowNetworkOffset<<<BLK_NUMS, BLK_DIM>>>( deviceGraph, flowNetwork, conComp, densestCore, finalCliqueData,   compCounter, upperBound,TOTAL_WARPS, totalComponents, k, max_lowerBound, totaLCliques);


    thrust::inclusive_scan(
        thrust::device_pointer_cast(flowNetwork.neighborOffset2),
        thrust::device_pointer_cast(flowNetwork.neighborOffset2 + flownetworkVertexSize),
        thrust::device_pointer_cast(flowNetwork.neighborOffset2));

    if(DEBUG){
        ui *h_offset2;
        h_offset2 = new ui[flownetworkVertexSize];

        chkerr(hipMemcpy(h_offset2,flowNetwork.neighborOffset2,(flownetworkVertexSize)*sizeof(ui),hipMemcpyDeviceToHost));

        cout<<"Neighbor Offset ";
        for(ui i=0;i<(flownetworkVertexSize);i++){
        cout<<h_offset2[i]<<" ";
        }
        cout<<endl;
    }
    createFlowNetwork<<<BLK_NUMS, BLK_DIM>>>( flowNetwork,  conComp,  densestCore,  finalCliqueData, compCounter,upperBound , TOTAL_WARPS, totalComponents, k, max_lowerBound , totaLCliques);
    hipDeviceSynchronize();

    if(DEBUG){

        ui *neigh_s;
        neigh_s = new ui[flownetworkNeighborSize];

        chkerr(hipMemcpy(neigh_s,flowNetwork.Edges,(flownetworkNeighborSize)*sizeof(ui),hipMemcpyDeviceToHost));

        cout<<"neighbor ";
        for(ui i=0;i<(flownetworkNeighborSize);i++){
        cout<<neigh_s[i]<<" ";
        }
        cout<<endl;
        double *f_capacity;

        f_capacity =  new double[flownetworkNeighborSize];


        chkerr(hipMemcpy(f_capacity,flowNetwork.capacity,(flownetworkNeighborSize)*sizeof(double),hipMemcpyDeviceToHost));

        cout<<"Capacity ";
        for(ui i=0; i<flownetworkNeighborSize; i++){
        cout<<f_capacity[i]<<" ";
        }
        cout<<endl;

        ui *height;
        double *excess;
        height = new ui[flownetworkVertexSize];
        excess = new double[flownetworkVertexSize];

        chkerr(hipMemcpy(height,flowNetwork.height, flownetworkVertexSize*sizeof(ui),hipMemcpyDeviceToHost));
        chkerr(hipMemcpy(excess,flowNetwork.excess, flownetworkVertexSize*sizeof(double),hipMemcpyDeviceToHost));

        for(ui i =0;i <flownetworkVertexSize; i++){
        cout<<height[i]<<" ";
        }cout<<endl;

        for(ui i =0; i<flownetworkVertexSize; i++){
        cout<<excess[i]<<" ";
        }cout<<endl;

    }

    ui *activeNodes;

    hipMalloc((void**)&activeNodes, partitionSize*TOTAL_WARPS * sizeof(ui));
    chkerr(hipMemset(flowNetwork.flow,0,(flownetworkNeighborSize)* sizeof(double)));

    size_t sharedMemorySize=  WARPS_EACH_BLK * sizeof(ui)+ WARPS_EACH_BLK *sizeof(ui);
    ui *componenetsLeft;
    ui left = 1;

    chkerr(hipMalloc((void**)&componenetsLeft,sizeof(ui)));
    chkerr(hipMemset(componenetsLeft,0,sizeof(ui)));


    double *densities;
    hipMalloc((void**)&densities,totalComponents*sizeof(double));
     chkerr(hipMemset(densities,0,totalComponents*sizeof(double)));

     //float *densityMax;

    //ui wajid =0;
    while(left!=0){
      chkerr(hipMemset(componenetsLeft,0,sizeof(ui)));
      pushRelabel<<<BLK_NUMS, BLK_DIM,sharedMemorySize>>>( flowNetwork,  conComp,  densestCore,  finalCliqueData, compCounter,upperBound,lowerBound, activeNodes,componenetsLeft,TOTAL_WARPS, totalComponents, k,totaLCliques, partitionSize);
      hipDeviceSynchronize();
      hipMemcpy(&left,componenetsLeft,sizeof(ui),hipMemcpyDeviceToHost);
      
      getResult<<<BLK_NUMS, BLK_DIM,sharedMemorySize>>>( flowNetwork,  conComp,  finalCliqueData, compCounter, upperBound, lowerBound, densities, TOTAL_WARPS, totalComponents, k, t);
      hipDeviceSynchronize();
      chkerr(hipMemset(flowNetwork.flow,0,(flownetworkNeighborSize)* sizeof(double)));


    }

    hipDeviceSynchronize();

    freeGraph(deviceGraph);


    return 0;
}