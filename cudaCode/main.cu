#include "./inc/common.h"
#include "./inc/graph.h"

#include "./utils/cuda_utils.cuh"
#include "./inc/gpuMemoryAllocation.cuh"
#include "./inc/helpers.cuh"

int main(int argc, const char * argv[]) {
    if (argc != 7) {
        cout << "Server wrong input parameters!" << endl;
        exit(1);
    }

    string filepath = argv[1]; // Path to the graph file. The graph should be represented as an adjacency list with space separators
    string motifPath = argv[2]; //Path to motif file. The motif should be represented as edge list with space sperators.
    ui k = atoi(argv[3]);
    ui pSize = atoi(argv[4]);
    ui cpSize = atoi(argv[5]);
    ui glBufferSize = atoi(argv[6]);

    Graph graph = Graph(filepath);

    //Motif M = Motif(motifPath);

    vector<ui> listingOrder;
    listingOrder.resize(graph.n);
    graph.getListingOrder(listingOrder);

    memoryAllocationGraph(deviceGraph, graph);
    memoryAllocationDAG(deviceDAG, graph.n, graph.m);

    // THIS PART IS RELATED TO GENERATING DAG

    ui *listOrder;
    chkerr(hipMalloc((void**)&(listOrder), graph.n * sizeof(ui)));
    chkerr(hipMemcpy(listOrder, listingOrder.data(), graph.n * sizeof(ui), hipMemcpyHostToDevice));

    // Get out degree in DAG
    generateDegreeDAG<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, deviceDAG, listOrder, graph.n, graph.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree of DAG");


    //copy out degree to offset
    chkerr(hipMemset(deviceDAG.offset, 0, sizeof(ui)));
    chkerr(hipMemcpy(deviceDAG.offset + 1, deviceDAG.degree, (graph.n) * sizeof(ui), hipMemcpyDeviceToDevice));

    // cummulative sum offset
    thrust::inclusive_scan(thrust::device_ptr<ui>(deviceDAG.offset), thrust::device_ptr<ui>(deviceDAG.offset + graph.n + 1), thrust::device_ptr<ui>(deviceDAG.offset));


    // Write neighbors of DAG
    size_t sharedMemoryGenDagNeig =  WARPS_EACH_BLK * sizeof(ui);
    generateNeighborDAG<<<BLK_NUMS, BLK_DIM,sharedMemoryGenDagNeig>>>(deviceGraph, deviceDAG, listOrder, graph.n, graph.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbor of DAG");

    chkerr(hipFree(listOrder));


    // THIS PART IS ABOUT CLIQUE LISTING ALGORITHM

    thrust::device_ptr<ui> dev_degree(deviceDAG.degree);


    auto max_iter = thrust::max_element(dev_degree, dev_degree + graph.n);
    int maxDegree = *max_iter;

    ui maxBitMask = memoryAllocationlevelData(levelData, k, pSize, cpSize, maxDegree, TOTAL_WARPS);

    int level = 0;
    int iterK = k;

    ui *labels;
    chkerr(hipMalloc((void**)&(labels), (graph.n * TOTAL_WARPS) * sizeof(ui)));
    thrust::device_ptr<ui> dev_labels(labels);
    thrust::fill(dev_labels, dev_labels + graph.n*TOTAL_WARPS, iterK);

    chkerr(hipMemcpy(deviceGraph.degree, graph.degree.data(), graph.n * sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemset(levelData.partialCliquesPartition, 0,  (TOTAL_WARPS * pSize)* sizeof(ui)));

    size_t sharedMemoryIntialClique =  WARPS_EACH_BLK * sizeof(ui);
    listIntialCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryIntialClique>>>(deviceDAG, levelData, labels, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Generate Intial Partial Cliques");
    ui partialSize = TOTAL_WARPS * pSize;
    //ui candidateSize = TOTAL_WARPS * cpSize;
    ui offsetSize = ((pSize / (k - 1)) + 1) * TOTAL_WARPS;

  
    ui offsetPartitionSize = ((pSize / (k-1)) + 1);
    createLevelDataOffset(levelData, offsetPartitionSize, TOTAL_WARPS);

    flushParitions<<<BLK_NUMS, BLK_DIM>>>(deviceDAG, levelData, pSize,cpSize,k, maxBitMask, level, TOTAL_WARPS);
    CUDA_CHECK_ERROR("Flush Partition data structure");

    iterK--;
    level++;

    int totalTasks;
    chkerr(hipMemcpy(&totalTasks, levelData.count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));
    size_t sharedMemoryMid =  WARPS_EACH_BLK * sizeof(ui);

    while(iterK > 2) {
        thrust::device_ptr<ui> dev_labels(labels);
        thrust::fill(dev_labels, dev_labels + graph.n*TOTAL_WARPS, iterK);
        chkerr(hipMemset(levelData.count, 0, (TOTAL_WARPS + 1) * sizeof(ui)));
        chkerr(hipMemset(levelData.temp, 0, (TOTAL_WARPS + 1) * sizeof(ui)));
        chkerr(hipMemset(levelData.offsetPartition, 0,  (offsetSize)* sizeof(ui)));
        chkerr(hipMemset(levelData.validNeighMaskPartition,0, (partialSize * maxBitMask) * sizeof(ui)));

        listMidCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryMid>>>(deviceDAG, levelData, labels, k, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, totalTasks, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Generate Mid Partial Cliques");

        createLevelDataOffset(levelData, offsetPartitionSize, TOTAL_WARPS);

        chkerr(hipMemset(levelData.offset,0,offsetSize*sizeof(ui)));
        chkerr(hipMemset(levelData.validNeighMask,0,partialSize*maxBitMask*sizeof(ui)));

        flushParitions<<<BLK_NUMS, BLK_DIM>>>(deviceDAG, levelData, pSize,cpSize,k, maxBitMask, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Flush Partition data structure");

        iterK--;
        level++;
    }

    ui t = 10; // Make it ud
    //TODO: decide the total number cliques and Free Level Data p 1
    chkerr(hipFree(labels));    
    memoryAllocationTrie(cliqueData, t, k);

    ui *totalCliques;
    chkerr(hipMalloc((void**)&totalCliques, sizeof(ui)));
    chkerr(hipMemset(totalCliques, 0, sizeof(ui)));
    size_t sharedMemoryFinal =  WARPS_EACH_BLK * sizeof(ui);


    chkerr(hipMemset(cliqueData.status, 0, t * sizeof(ui)));
    chkerr(hipMemset(cliqueData.trie, 0, t * k * sizeof(ui)));
    if(iterK == 2) {
        writeFinalCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryFinal>>>(deviceGraph, deviceDAG, levelData, cliqueData, totalCliques, k, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, t,totalTasks, level, TOTAL_WARPS);
        CUDA_CHECK_ERROR("Generate Full Cliques");
    }


    freeLevelData(levelData);
    freeLevelPartitionData(levelData);
    freeDAG(deviceDAG);

    ui tt;
    chkerr(hipMemcpy(&tt, totalCliques, sizeof(ui), hipMemcpyDeviceToHost));
    cout<<endl<<"total cliques "<<tt<<endl;

    size_t sharedMemorySort =  2*k*WARPS_EACH_BLK * sizeof(ui);
    sortTrieData<<<BLK_NUMS, BLK_DIM,sharedMemorySort>>>(deviceGraph, cliqueData, tt,t, k, TOTAL_THREAD);
    CUDA_CHECK_ERROR("Sort Trie Data Structure");

    int *h_cliques,*status;
    h_cliques = new int[t*k];
    status = new int[t];
    chkerr(hipMemcpy(h_cliques, cliqueData.trie, k * t * sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(status, cliqueData.status, t * sizeof(ui), hipMemcpyDeviceToHost));
    cout<<endl;
    for(int i =0;i<k;i++){
      cout<<endl<<"CL "<<i<<"  ";
      for(int j =0;j<t;j++){
        cout<<h_cliques[i*t+j]<<" ";
      }
    }
    cout<<endl<<"stat  ";
    for(int i = 0; i < t; i++) {
        cout << status[i] << " ";

    }

    ui *h_cdegree;
    h_cdegree = new ui[graph.n];
    chkerr(hipMemcpy(h_cdegree, deviceGraph.cliqueDegree, graph.n* sizeof(ui), hipMemcpyDeviceToHost));
    cout<<endl;
    for(int i = 0; i < graph.n; i++) {
        cout <<i<<" ";
    } 
    cout<<endl;
    for(int i = 0; i < graph.n; i++) {
        cout << h_cdegree[i] << " ";
    }

    //TODO:  CLIQUE CORE DECOMPOSE

    level = 0;
    ui count = 0;
    ui *globalCount = NULL;
    ui *bufTails  = NULL;
    ui *glBuffers = NULL;

    chkerr(hipMalloc((void**)&(globalCount), sizeof(ui)));
    chkerr(hipMalloc((void**)&(bufTails), BLK_NUMS*sizeof(ui)));
    chkerr(hipMalloc((void**)&(glBuffers), BLK_NUMS*glBufferSize*sizeof(ui)));
    chkerr(hipMemset(globalCount, 0, sizeof(ui)));
    chkerr(hipMalloc(&glBuffers,BLK_NUMS*glBufferSize*sizeof(ui)));
    hipDeviceSynchronize();

//    chkerr(hipMemset(glBuffers, 0, BLK_NUMS*glBufferSize*sizeof(ui)));

    chkerr(hipMemcpy(deviceGraph.cliqueCore, deviceGraph.cliqueDegree, graph.n * sizeof(ui), hipMemcpyDeviceToDevice));
    while(count < graph.n){
        hipMemset(bufTails, 0, sizeof(unsigned int)*BLK_NUMS);

        // Select nodes whoes current degree is level, that means they should be removed as part of the level core 
        selectNodes<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, bufTails, glBuffers, glBufferSize, graph.n, level);
        hipDeviceSynchronize();
        
        //Total number of verticies in buffer
        thrust::device_vector<ui> dev_vec(bufTails, bufTails + BLK_NUMS);
        ui sum = thrust::reduce(dev_vec.begin(), dev_vec.end(), 0, thrust::plus<ui>());

        //Bases on total vertices device to either use Warp or Block to process one vertex and its cliques
        if(sum > 2* BLK_NUMS){
            processNodesByWarp<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, cliqueData , bufTails, glBuffers, globalCount, glBufferSize, graph.n, level, k, tt);
            hipDeviceSynchronize();
        }else{
            processNodesByBlock<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, cliqueData , bufTails, glBuffers, globalCount, glBufferSize, graph.n, level, k, tt);

        }

        chkerr(hipMemcpy(&count, globalCount, sizeof(unsigned int), hipMemcpyDeviceToHost));    
        level++;
    }
    graph.kmax = level-1;


    //TODO: LOCATE CORE

    //TODO: LISTING AGAIN

    //TODO: EDGE PRUNING

    //TODO: COMPONENT DECOMPOSE

    //TODO: DYNAMIC CORE EXACT
    freTrie(cliqueData);
    freeGraph(deviceGraph);
    return 0;
}