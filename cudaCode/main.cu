#include "hip/hip_runtime.h"
#include "./inc/common.h"
#include "./inc/graph.h"

#include "./utils/cuda_utils.cuh"
#include "./inc/gpuMemoryAllocation.cuh"
#include "./inc/helpers.cuh"
#include <thrust/count.h>



void generateDAG(const Graph& graph,deviceGraphPointers& deviceGraph,deviceDAGpointer& deviceDAG , vector<ui> listingOrder){
    
    memoryAllocationDAG(deviceDAG, graph.n, graph.m);

    ui *listOrder;
    chkerr(hipMalloc((void**)&(listOrder), graph.n * sizeof(ui)));
    chkerr(hipMemcpy(listOrder, listingOrder.data(), graph.n * sizeof(ui), hipMemcpyHostToDevice));

    // Get out degree in DAG
    generateDegreeDAG<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, deviceDAG, listOrder, graph.n, graph.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree of DAG");


    //copy out degree to offset
    chkerr(hipMemset(deviceDAG.offset, 0, sizeof(ui)));
    chkerr(hipMemcpy(deviceDAG.offset + 1, deviceDAG.degree, (graph.n) * sizeof(ui), hipMemcpyDeviceToDevice));

    // cummulative sum offset
    thrust::inclusive_scan(thrust::device_ptr<ui>(deviceDAG.offset), thrust::device_ptr<ui>(deviceDAG.offset + graph.n + 1), thrust::device_ptr<ui>(deviceDAG.offset));


    // Write neighbors of DAG
    size_t sharedMemoryGenDagNeig =  WARPS_EACH_BLK * sizeof(ui);
    generateNeighborDAG<<<BLK_NUMS, BLK_DIM,sharedMemoryGenDagNeig>>>(deviceGraph, deviceDAG, listOrder, graph.n, graph.m, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbor of DAG");

    chkerr(hipFree(listOrder));

}

ui listAllCliques(const Graph& graph,deviceGraphPointers& deviceGraph,deviceDAGpointer& deviceDAG,cliqueLevelDataPointer levelData, ui k, ui pSize,ui  cpSize, ui t){
    thrust::device_ptr<ui> dev_degree(deviceDAG.degree);


    auto max_iter = thrust::max_element(dev_degree, dev_degree + graph.n);
    int maxDegree = *max_iter;

    ui maxBitMask = memoryAllocationlevelData(levelData, k, pSize, cpSize, maxDegree, TOTAL_WARPS);

    int level = 0;
    int iterK = k;

    ui *labels;
    chkerr(hipMalloc((void**)&(labels), (graph.n * TOTAL_WARPS) * sizeof(ui)));
    thrust::device_ptr<ui> dev_labels(labels);
    thrust::fill(dev_labels, dev_labels + graph.n*TOTAL_WARPS, iterK);

    chkerr(hipMemcpy(deviceGraph.degree, graph.degree.data(), graph.n * sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMemset(levelData.partialCliquesPartition, 0,  (TOTAL_WARPS * pSize)* sizeof(ui)));

    size_t sharedMemoryIntialClique =  WARPS_EACH_BLK * sizeof(ui);
    listIntialCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryIntialClique>>>(deviceDAG, levelData, labels, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, level, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Intial Partial Cliques");
    ui partialSize = TOTAL_WARPS * pSize;
    //ui candidateSize = TOTAL_WARPS * cpSize;
    ui offsetSize = ((pSize / (k - 1)) + 1) * TOTAL_WARPS;

  
    ui offsetPartitionSize = ((pSize / (k-1)) + 1);
    createLevelDataOffset(levelData, offsetPartitionSize, TOTAL_WARPS);

    flushParitions<<<BLK_NUMS, BLK_DIM>>>(deviceDAG, levelData, pSize,cpSize,k, maxBitMask, level, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Flush Partition data structure");

    iterK--;
    level++;

    int totalTasks;
    chkerr(hipMemcpy(&totalTasks, levelData.count + TOTAL_WARPS, sizeof(ui), hipMemcpyDeviceToHost));
    size_t sharedMemoryMid =  WARPS_EACH_BLK * sizeof(ui);

    while(iterK > 2) {
        thrust::device_ptr<ui> dev_labels(labels);
        thrust::fill(dev_labels, dev_labels + graph.n*TOTAL_WARPS, iterK);
        chkerr(hipMemset(levelData.count, 0, (TOTAL_WARPS + 1) * sizeof(ui)));
        chkerr(hipMemset(levelData.temp, 0, (TOTAL_WARPS + 1) * sizeof(ui)));
        chkerr(hipMemset(levelData.offsetPartition, 0,  (offsetSize)* sizeof(ui)));
        chkerr(hipMemset(levelData.validNeighMaskPartition,0, (partialSize * maxBitMask) * sizeof(ui)));

        listMidCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryMid>>>(deviceDAG, levelData, labels, k, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, totalTasks, level, TOTAL_WARPS);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("Generate Mid Partial Cliques");

        createLevelDataOffset(levelData, offsetPartitionSize, TOTAL_WARPS);

        chkerr(hipMemset(levelData.offset,0,offsetSize*sizeof(ui)));
        chkerr(hipMemset(levelData.validNeighMask,0,partialSize*maxBitMask*sizeof(ui)));

        flushParitions<<<BLK_NUMS, BLK_DIM>>>(deviceDAG, levelData, pSize,cpSize,k, maxBitMask, level, TOTAL_WARPS);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("Flush Partition data structure");

        iterK--;
        level++;
    }

    //ui t = 10; // Make it ud
    //TODO: decide the total number cliques and Free Level Data p 1
    chkerr(hipFree(labels));    
    memoryAllocationTrie(cliqueData, t, k);

    ui *totalCliques;
    chkerr(hipMalloc((void**)&totalCliques, sizeof(ui)));
    chkerr(hipMemset(totalCliques, 0, sizeof(ui)));
    size_t sharedMemoryFinal =  WARPS_EACH_BLK * sizeof(ui);


    thrust::device_ptr<int> dev_ptr(cliqueData.status);

    thrust::fill(dev_ptr, dev_ptr + t, -2);
    chkerr(hipMemset(cliqueData.trie, 0, t * k * sizeof(ui)));
    if(iterK == 2) {
        writeFinalCliques<<<BLK_NUMS, BLK_DIM,sharedMemoryFinal>>>(deviceGraph, deviceDAG, levelData, cliqueData, totalCliques, k, iterK, graph.n, graph.m, pSize, cpSize, maxBitMask, t,totalTasks, level, TOTAL_WARPS);
        hipDeviceSynchronize();
        CUDA_CHECK_ERROR("Generate Full Cliques");
    }


    freeLevelData(levelData);
    freeLevelPartitionData(levelData);
    freeDAG(deviceDAG);

    ui tt;
    chkerr(hipMemcpy(&tt, totalCliques, sizeof(ui), hipMemcpyDeviceToHost));
    cout<<endl<<"total cliques "<<tt<<endl;

    // Sort Not needed
    //size_t sharedMemorySort =  2*k*WARPS_EACH_BLK * sizeof(ui);
    //sortTrieData<<<BLK_NUMS, BLK_DIM,sharedMemorySort>>>(deviceGraph, cliqueData, tt,t, k, TOTAL_THREAD);
    //hipDeviceSynchronize();
    //CUDA_CHECK_ERROR("Sort Trie Data Structure");

    hipFree(totalCliques);
    return tt;
}

void cliqueCoreDecompose(const Graph& graph,deviceGraphPointers& deviceGraph,deviceCliquesPointer& cliqueData, ui &maxCore, double &maxDensity, ui &coreSize, ui &coreTotalCliques, ui glBufferSize, ui k, ui t, ui tt){
    ui level = 0;
    ui count = 0;
    ui *globalCount = NULL;
    ui *bufTails  = NULL;
    ui *glBuffers = NULL;

    chkerr(hipMalloc((void**)&(globalCount), sizeof(ui)));
    chkerr(hipMalloc((void**)&(bufTails), BLK_NUMS*sizeof(ui)));
    chkerr(hipMalloc((void**)&(glBuffers), BLK_NUMS*glBufferSize*sizeof(ui)));
    chkerr(hipMemset(globalCount, 0, sizeof(ui)));
    hipDeviceSynchronize();



    chkerr(hipMemcpy(deviceGraph.cliqueCore, deviceGraph.cliqueDegree, graph.n * sizeof(ui), hipMemcpyDeviceToDevice));


    thrust::device_vector<int> dev_vec(cliqueData.status, cliqueData.status + t);

    ui currentCliques = thrust::count(dev_vec.begin(), dev_vec.end(), -1);
    double currentDensity = static_cast<double>(currentCliques) / (graph.n - count);

    maxDensity = currentDensity;
    maxCore = 0;
    coreTotalCliques = currentCliques;
    coreSize = graph.n;


    while(count < graph.n){
        hipMemset(bufTails, 0, sizeof(ui)*BLK_NUMS);

        // Select nodes whoes current degree is level, that means they should be removed as part of the level core
        selectNodes<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, bufTails, glBuffers, glBufferSize, graph.n, level);
        hipDeviceSynchronize();

        //Total number of verticies in buffer
        thrust::device_vector<ui> dev_vec1(bufTails, bufTails + BLK_NUMS);
        ui sum = thrust::reduce(dev_vec1.begin(), dev_vec1.end(), 0, thrust::plus<ui>());
        cout<<"sum "<<sum;
        hipDeviceSynchronize();



        processNodesByWarp<<<BLK_NUMS, BLK_DIM>>>(deviceGraph, cliqueData , bufTails, glBuffers, globalCount, glBufferSize, graph.n, level, k, t, tt);
        hipDeviceSynchronize();

        //debug
        chkerr(hipMemcpy(h_coreCliques, deviceGraph.cliqueCore, graph.n * sizeof(int), hipMemcpyDeviceToHost));
        cout<<endl<<"Core Cliques ";
        for(int i = 0; i < graph.n; i++){
            cout<<h_coreCliques[i]<<" ";
        }
        cout<<endl;
        //debug

        chkerr(hipMemcpy(&count, globalCount, sizeof(unsigned int), hipMemcpyDeviceToHost));

        level++;
        thrust::device_vector<int> dev_vec2(cliqueData.status, cliqueData.status + t);


       if((graph.n - count)!=0){
        currentCliques = thrust::count(dev_vec2.begin(), dev_vec2.end(), -1);
        currentDensity = static_cast<double>(currentCliques) / (graph.n - count);

        if(currentDensity>=maxDensity){
            maxDensity = currentDensity;
            maxCore = level;
            coreTotalCliques = currentCliques;
            coreSize = graph.n-count;

        }
        
    

       }

        hipDeviceSynchronize();

    }
    hipFree(globalCount);
    hipFree(bufTails);
    hipFree(glBuffers);

}


ui generateDensestCore(const Graph& graph,deviceGraphPointers& deviceGraph, densestCorePointer &densestCore, ui *reverseMap, ui coreSize, ui coreTotalCliques, ui lowerBoundDensity){
    memoryAllocationDensestCore(densestCore, coreSize, lowerBoundDensity , coreTotalCliques);
    ui *globalCount;

    chkerr(hipMalloc((void**)&globalCount, sizeof(ui)));
    chkerr(hipMemset(globalCount, 0, sizeof(ui)));


    generateDensestCore<<<BLK_NUMS, BLK_DIM>>>(deviceGraph,densestCore,globalCount,graph.n,lowerBoundDensity,TOTAL_WARPS);
    hipDeviceSynchronize();

    thrust::inclusive_scan(thrust::device_ptr<ui>(densestCore.offset), thrust::device_ptr<ui>(densestCore.offset + coreSize + 1), thrust::device_ptr<ui>(densestCore.offset));

    ui edgeCountCore;
    chkerr(hipMemcpy(&edgeCountCore, densestCore.offset+coreSize , sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMalloc((void**)&(densestCore.neighbors), edgeCountCore * sizeof(ui)));

    thrust::device_ptr<unsigned int> d_vertex_map_ptr(densestCore.mapping);

    thrust::device_ptr<unsigned int> d_reverse_map_ptr(reverseMap);

    thrust::device_vector<unsigned int> d_indices(coreSize);

    thrust::sequence(d_indices.begin(), d_indices.end());

    // Scatter indices into the reverse mapping array

    thrust::scatter(d_indices.begin(), d_indices.end(), d_vertex_map_ptr, d_reverse_map_ptr);

    size_t sharedMemoryGenNeighCore =  WARPS_EACH_BLK * sizeof(ui);
    generateNeighborDensestCore<<<BLK_NUMS, BLK_DIM,sharedMemoryGenNeighCore>>>(deviceGraph,densestCore,reverseMap,lowerBoundDensity,TOTAL_WARPS);
    hipDeviceSynchronize();

    return edgeCountCore;

}

ui prune(densestCorePointer &densestCore, deviceCliquesPointer &cliqueData, ui *pruneStatus, ui *reverseMap, ui *newOffset, ui *newNeighbors, ui vertexCount, ui edgecount, ui k, ui t, ui t, ui lowerBoundDensity){
    
    //Prune
    chkerr(hipMalloc((void**)&pruneStatus, edgecount * sizeof(ui)));

    thrust::device_ptr<ui> d_pruneStatus(pruneStatus);

    // Fill the array with 1 using Thrust
    thrust::fill(d_pruneStatus, d_pruneStatus + edgecount, 1);

    pruneEdges<<<BLK_NUMS, BLK_DIM>>>( densestCore,  cliqueData,reverseMap, pruneStatus, t, tt,  k, lowerBoundDensity);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Get prune status of each edge");


    // Get out degree after prune
    chkerr(hipMalloc((void**)&newOffset, (vertexCount+1) * sizeof(ui)));
    chkerr(hipMemset(newOffset, (vertexCount+1)  , sizeof(ui)));


    generateDegreeAfterPrune<<<BLK_NUMS, BLK_DIM>>>(densestCore , pruneStatus, newOffset, vertexCount , edgecount, TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Degree after pruning");

    // cummulative sum offset
    thrust::inclusive_scan(thrust::device_ptr<ui>(newOffset), thrust::device_ptr<ui>(newOffset + vertexCount + 1), thrust::device_ptr<ui>(newOffset));

    ui newEdgeCount;
    chkerr(hipMemcpy(&newEdgeCount, newOffset+ vertexCount , sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMalloc((void**)&(newNeighbors), newEdgeCount * sizeof(ui)));

    // Write neighbors of after output
    size_t sharedMemoryGenNeig =  WARPS_EACH_BLK * sizeof(ui);
    generateNeighborAfterPrune<<<BLK_NUMS, BLK_DIM,sharedMemoryGenNeig>>>(densestCore , pruneStatus, newOffset, newNeighbors, vertexCount, edgecount , TOTAL_WARPS);
    hipDeviceSynchronize();
    CUDA_CHECK_ERROR("Generate Neighbor after prune");

    return newEdgeCount;
    
}


int main(int argc, const char * argv[]) {
    if (argc != 7) {
        cout << "Server wrong input parameters!" << endl;
        exit(1);
    }

    string filepath = argv[1]; // Path to the graph file. The graph should be represented as an adjacency list with space separators
    string motifPath = argv[2]; //Path to motif file. The motif should be represented as edge list with space sperators.
    ui k = atoi(argv[3]);
    ui pSize = atoi(argv[4]);
    ui cpSize = atoi(argv[5]);
    ui glBufferSize = atoi(argv[6]);

    // Need better way to do this
    ui t=10;

    //Debug
    cout << "filepath: " << filepath << endl;
    cout << "motifPath: " << motifPath << endl;
    cout <<"k: " << k << endl;
    cout << "pSize: " << pSize << endl;
    cout << "cpSize: " << cpSize << endl;

    Graph graph = Graph(filepath);

    //Motif M = Motif(motifPath);

    vector<ui> listingOrder;
    listingOrder.resize(graph.n);
    graph.getListingOrder(listingOrder);

    memoryAllocationGraph(deviceGraph, graph);

    // GENERATES DAG
    generateDAG(graph, deviceGraph, deviceDAG,listingOrder);

    // CLIQUE LISTING ALGORITHM
    ui tt = listAllCliques(graph, deviceGraph, deviceDAG, levelData, k, pSize, cpSize,t);


    // Debug start 
    int *h_cliques,*status;
    h_cliques = new int[t*k];
    status = new int[t];
    chkerr(hipMemcpy(h_cliques, cliqueData.trie, k * t * sizeof(ui), hipMemcpyDeviceToHost));
    chkerr(hipMemcpy(status, cliqueData.status, t * sizeof(ui), hipMemcpyDeviceToHost));
    cout<<endl;
    for(int i =0;i<k;i++){
      cout<<endl<<"CL "<<i<<"  ";
      for(int j =0;j<t;j++){
        cout<<h_cliques[i*t+j]<<" ";
      }
    }
    cout<<endl<<"stat  ";
    for(int i = 0; i < t; i++) {
        cout << status[i] << " ";

    }

    ui *h_cdegree;
    h_cdegree = new ui[graph.n];
    chkerr(hipMemcpy(h_cdegree, deviceGraph.cliqueDegree, graph.n* sizeof(ui), hipMemcpyDeviceToHost));
    cout<<endl;
    for(int i = 0; i < graph.n; i++) {
        cout <<i<<" ";
    } 
    cout<<endl;
    for(int i = 0; i < graph.n; i++) {
        cout << h_cdegree[i] << " ";
    }

    //Debug end

    // CLIQUE CORE DECOMPOSE
    ui coreSize, coreTotalCliques,maxCore;
    double maxDensity;
    cout<<endl<<"BEFORE DECOMPOSE"<<endl;
    cliqueCoreDecompose(graph,deviceGraph,cliqueData,maxCore, maxDensity, coreSize, coreTotalCliques,glBufferSize, k,  t, tt);

    //LOCATE CORE
    ui *reverseMap;
    chkerr(hipMalloc((void**)&reverseMap, graph.n * sizeof(ui)));
    hipMemset(reverseMap, 0xFF, graph.n * sizeof(ui));
    ui lowerBoundDensity = static_cast<ui>(std::ceil(maxDensity));

    ui edgecount = generateDensestCore(graph,deviceGraph,  densestCore, reverseMap, coreSize, coreTotalCliques,lowerBoundDensity);

    //TODO: LISTING AGAIN not need added level as status of each clique to track the cores


    //TODO: EDGE PRUNING
    ui *pruneStatus;
    ui *newOffset;
    ui *newNeighbors;
    ui vertexCount;
    chkerr(hipMemcpy(&vertexCount, densestCore.n, sizeof(ui), hipMemcpyDeviceToHost));

    
    ui newEdgeCount = prune(densestCore, cliqueData, pruneStatus, reverseMap, newOffset, newNeighbors, vertexCount, edgecount, k, t, t, lowerBoundDensity);

    //TODO: COMPONENT DECOMPOSE

    //TODO: DYNAMIC CORE EXACT
    freTrie(cliqueData);
    freeGraph(deviceGraph);
    return 0;
}