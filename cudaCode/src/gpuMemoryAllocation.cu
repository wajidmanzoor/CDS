#include "hip/hip_runtime.h"
#include "../utils/cuda_utils.cuh"
#include "../inc/gpuMemoryAllocation.cuh"


void memoryAllocationGraph(deviceGraphPointers &G, Graph &graph) {
    ui n = graph.n;
    ui m = graph.m;
    chkerr(hipMalloc((void**)&(G.offset), (n + 1) * sizeof(ui)));
    chkerr(hipMemcpy(G.offset, graph.offset.data(), (n + 1) * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.neighbors), (2 * m) * sizeof(ui)));
    chkerr(hipMemcpy(G.neighbors, graph.neighbors.data(), (2 * m) * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.degree), n * sizeof(ui)));
    chkerr(hipMemcpy(G.degree, graph.degree.data(), n * sizeof(ui), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(G.cliqueDegree), n * sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.cliqueCore), n * sizeof(int)));

    chkerr(hipMalloc((void**)&(G.cliqueCorePeelSequence), n * sizeof(ui)));
    chkerr(hipMalloc((void**)&(G.density), n * sizeof(double)));
    chkerr(hipMalloc((void**)&(G.motifCount), n * sizeof(ui)));

    hipDeviceSynchronize();
}

void memoryAllocationDAG(deviceDAGpointer &D, ui n, ui m) {
    chkerr(hipMalloc((void**)&(D.offset), (n + 1) * sizeof(ui)));
    chkerr(hipMemset(D.offset, 0, (n + 1) * sizeof(ui)));

    chkerr(hipMalloc((void**)&(D.neighbors), m * sizeof(ui)));
    chkerr(hipMalloc((void**)&(D.degree), n * sizeof(ui)));
    hipDeviceSynchronize();
}

/*void memoryAllocationMotif(deviceMotifPointers &M, Motif &motif) {
    ui n = motif.size;

    chkerr(hipMalloc((void**)&(M.adjacencyMatrix), (n * n) * sizeof(ui)));

    // TODO: Flatten motif.adjMatrix
    std::vector<ui> flatMatrix(n * n);
    // Flatten the 2D adjacency matrix into a 1D array
    for (ui i = 0; i < n; ++i) {
        for (ui j = 0; j < n; ++j) {
            flatMatrix[i * n + j] = (*motif.adjMatrix)[i][j];
        }
    }

    chkerr(hipMemcpy(M.adjacencyMatrix, flatMatrix.data(), (n * n) * sizeof(ui), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}*/

void memoryAllocationComponent(deviceComponentPointers &C, ui n, ui m) {
    chkerr(hipMalloc((void**)&(C.componentOffset), (n + 1) * sizeof(ui)));
    chkerr(hipMemset(C.componentOffset, 0, (n + 1) * sizeof(ui)));
    chkerr(hipMalloc((void**)&(C.components), n * sizeof(ui)));
    chkerr(hipMalloc((void**)&(C.mapping), n * sizeof(ui)));
    hipDeviceSynchronize();
}

void memoryAllocationresult(deviceResultpointer &R, ui n) {
    chkerr(hipMalloc((void**)&(R.maxDensity), sizeof(double)));
    chkerr(hipMalloc((void**)&(R.numVertex), sizeof(ui)));
    chkerr(hipMalloc((void**)&(R.component), sizeof(ui)));
    chkerr(hipMalloc((void**)&(R.status), n * sizeof(ui)));
    hipDeviceSynchronize();
}

void memoryAllocationTrie(deviceCliquesPointer &C, ui t, ui k) {
    chkerr(hipMalloc((void**)&(C.trie), (t * k) * sizeof(ui)));
    chkerr(hipMalloc((void**)&(C.status), t * sizeof(int)));
    hipDeviceSynchronize();
}

ui memoryAllocationlevelData(cliqueLevelDataPointer &L, ui k, ui pSize, ui cpSize, ui maxDegree, ui totalWarps) {
    ui partialSize = totalWarps * pSize;
    ui candidateSize = totalWarps * cpSize;
    ui offsetSize = ((pSize / (k - 1)) + 1) * totalWarps;
    ui maxBitMask = (maxDegree + 31) / 32;
    ui maskSize = (cpSize * maxBitMask) * totalWarps;
    ui max_ = partialSize / (k - 1);

    chkerr(hipMalloc((void**)&(L.partialCliquesPartition), partialSize * sizeof(ui)));
    chkerr(hipMalloc((void**)&(L.partialCliques), partialSize * sizeof(ui)));

    chkerr(hipMalloc((void**)&(L.candidatesPartition), candidateSize * sizeof(ui)));
    chkerr(hipMalloc((void**)&(L.candidates), candidateSize * sizeof(ui)));

    chkerr(hipMalloc((void**)&(L.validNeighMaskPartition), maskSize * sizeof(ui)));
    chkerr(hipMalloc((void**)&(L.validNeighMask), maskSize * sizeof(ui)));

    chkerr(hipMemset(L.validNeighMask, 0, maskSize * sizeof(ui)));
    chkerr(hipMemset(L.validNeighMaskPartition, 0, maskSize * sizeof(ui)));

    chkerr(hipMalloc((void**)&(L.offsetPartition), offsetSize * sizeof(ui)));
    chkerr(hipMemset(L.offsetPartition, 0, offsetSize * sizeof(ui)));

    chkerr(hipMalloc((void**)&(L.offset), offsetSize * sizeof(ui)));
    chkerr(hipMemset(L.offset, 0, offsetSize * sizeof(ui)));

    chkerr(hipMalloc((void**)&(L.count), (totalWarps + 1) * sizeof(ui)));
    chkerr(hipMalloc((void**)&(L.temp), (totalWarps + 1) * sizeof(ui)));
    chkerr(hipMemset(L.temp, 0, (totalWarps + 1) * sizeof(ui)));
    chkerr(hipMemset(L.count, 0, (totalWarps + 1) * sizeof(ui)));
    chkerr(hipMalloc((void**)&(L.max), sizeof(ui)));
    chkerr(hipMemcpy(L.max, &max_, sizeof(ui), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    return maxBitMask;
}
void memoryAllocationDensestCore(densestCorePointer &C, ui n, ui density, ui totalCliques){


    chkerr(hipMalloc((void**)&(C.mapping), n* sizeof(ui)));

    chkerr(hipMalloc((void**)&(C.offset), (n+1) * sizeof(ui)));
    chkerr(hipMemset(C.offset, 0, (n+1)* sizeof(ui)));

    //neighbors will be allocated once we now the size

    chkerr(hipMalloc((void**)&(C.cliqueDegree), n * sizeof(ui)));
    //chkerr(hipMalloc((void**)&(C.cliqueCore), n * sizeof(ui)));

    chkerr(hipMalloc((void**)&(C.density), n * sizeof(double)));
    chkerr(hipMemcpy(C.density, &density, sizeof(double), hipMemcpyHostToDevice));

    chkerr(hipMalloc((void**)&(C.n), sizeof(ui)));
    chkerr(hipMemcpy(C.n, &n, sizeof(ui), hipMemcpyHostToDevice));
    chkerr(hipMalloc((void**)&(C.m), sizeof(ui)));
    
    chkerr(hipMalloc((void**)&(C.totalCliques), sizeof(ui)));
    chkerr(hipMemcpy(C.totalCliques, &totalCliques, sizeof(ui), hipMemcpyHostToDevice));  
    chkerr(hipMalloc((void**)&(C.reverseMap), n * sizeof(ui)));
    hipDeviceSynchronize();

}


// Memory deallocation functions
void freeGraph(deviceGraphPointers &G) {
    chkerr(hipFree(G.offset));
    chkerr(hipFree(G.neighbors));
    chkerr(hipFree(G.degree));
    chkerr(hipFree(G.cliqueDegree));
    chkerr(hipFree(G.cliqueCore));
    chkerr(hipFree(G.cliqueCorePeelSequence));
    chkerr(hipFree(G.density));
    chkerr(hipFree(G.motifCount));
}

void memoryAllocationPrunnedNeighbors(devicePrunedNeighbors &prunedNeighbors, ui n, ui m){
    chkerr(hipMalloc((void**)&(prunedNeighbors.newOffset), (n+1) * sizeof(ui)));
    chkerr(hipMemset(prunedNeighbors.newOffset, 0, (n+1)* sizeof(ui)));

    chkerr(hipMalloc((void**)&(prunedNeighbors.pruneStatus), (2 * m) * sizeof(ui)));
    hipDeviceSynchronize();
}

void memoryAllocationFlowNetwork(deviceFlowNetworkPointers &flowNetwork, ui size){
    chkerr(hipMalloc((void**)&(flowNetwork.toEdge), size * sizeof(ui)));

    chkerr(hipMalloc((void**)&(flowNetwork.capacity), size * sizeof(capacity)));
    chkerr(hipMalloc((void**)&(flowNetwork.flow), size* sizeof(capacity)));

    hipDeviceSynchronize();

}

/*void freeMotif(deviceMotifPointers &M) {
    chkerr(hipFree(M.adjacencyMatrix));
}*/


void freeComponents(deviceComponentPointers &C) {
    chkerr(hipFree(C.componentTotal));
    chkerr(hipFree(C.componentOffset));
    chkerr(hipFree(C.components));
    chkerr(hipFree(C.mapping));

}

void freeResults(deviceResultpointer &R) {
    chkerr(hipFree(R.maxDensity));
    chkerr(hipFree(R.numVertex));
    chkerr(hipFree(R.component));
    chkerr(hipFree(R.status));
}

void freTrie(deviceCliquesPointer &C) {
    chkerr(hipFree(C.trie));
    chkerr(hipFree(C.status));
}

void freeDAG(deviceDAGpointer &D) {
    chkerr(hipFree(D.offset));
    chkerr(hipFree(D.neighbors));
    chkerr(hipFree(D.degree));
}

void freeLevelPartitionData(cliqueLevelDataPointer &L) {
    chkerr(hipFree(L.partialCliquesPartition));
    chkerr(hipFree(L.candidatesPartition));
    chkerr(hipFree(L.offsetPartition));
    chkerr(hipFree(L.validNeighMaskPartition));
    chkerr(hipFree(L.temp));
}

void freeLevelData(cliqueLevelDataPointer &L) {
    chkerr(hipFree(L.partialCliques));
    chkerr(hipFree(L.candidates));
    chkerr(hipFree(L.offset));
    chkerr(hipFree(L.validNeighMask));
    chkerr(hipFree(L.count));
    chkerr(hipFree(L.max));
}

void freeDensestCore(densestCorePointer &C){
    chkerr(hipFree(C.mapping));
    chkerr(hipFree(c.reverseMap));
    chkerr(hipFree(C.offset));
    chkerr(hipFree(C.neighbors));
    chkerr(hipFree(C.density));
    chkerr(hipFree(C.n));
    chkerr(hipFree(C.m));
    chkerr(hipFree(C.totalCliques));
    chkerr(hipFree(C.cliqueDegree));
 
}

void freePrunnedNeighbors(devicePrunedNeighbors &prunedNeighbors){
    chkerr(hipFree(prunedNeighbors.newOffset));
    chkerr(hipFree(prunedNeighbors.newNeighbors));
    chkerr(hipFree(prunedNeighbors.pruneStatus));
}

void freeFlowNetwork(deviceFlowNetworkPointers &flowNetwork){
    chkerr(hipFree(flowNetwork.toEdge));
    chkerr(hipFree(flowNetwork.capacity));
    chkerr(hipFree(flowNetwork.flow));
    hipDeviceSynchronize();

}

