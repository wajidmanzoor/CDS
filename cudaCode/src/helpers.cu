#include "hip/hip_runtime.h"
#include "../utils/cuda_utils.cuh"
#include "../inc/helpers.cuh"

//TODO: remove unused variables 
//TODO: Make code a bit more structured and clean
//TODO: Try to find more ways to optimize

__device__ double fact(ui k){
    double res = 1;
    int i = k;
    while(i>1){
        res= res*i;
        i--;

    }
    return res;
}

__device__ double power(ui totalCliques, double p) {
    return powf((double)totalCliques, p );
}

__global__ void generateDegreeDAG(deviceGraphPointers G, deviceDAGpointer D, ui *listingOrder, ui n, ui m, ui totalWarps) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < n; i += totalWarps) {
        ui start = G.offset[i];
        ui end = G.offset[i+1];
        ui total = end - start;
        ui neigh;
        int count = 0;
        for(int j = laneId; j < total; j += warpSize) {
            neigh = G.neighbors[start + j];
            if(listingOrder[i] < listingOrder[neigh]) {
                count++;
            }
        }
        
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            count += __shfl_down_sync(0xFFFFFFFF, count, offset);
        }

        if(laneId == 0) {
            D.degree[i] = count;
        }
    }
}

__global__ void generateNeighborDAG(deviceGraphPointers G, deviceDAGpointer D, ui *listingOrder, ui n, ui m, ui totalWarps) {

    extern __shared__ char sharedMemory[];
    ui sizeOffset = 0;

    ui *counter = (ui *)(sharedMemory + sizeOffset);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < n; i += totalWarps) {
        if(laneId==0){
          counter[threadIdx.x / warpSize] = D.offset[i];
        }
        __syncwarp();
        ui start = G.offset[i];
        ui end = G.offset[i+1];
        ui total = end - start;
        ui neigh;
        for(int j = laneId; j < total; j += warpSize) {
            neigh = G.neighbors[start + j];

            if(listingOrder[i] < listingOrder[neigh]) {
                int loc = atomicAdd(&counter[threadIdx.x / warpSize], 1);
                D.neighbors[loc] = neigh;

            }
        }
      __syncwarp();
    }
}

__global__ void listIntialCliques(deviceDAGpointer D, cliqueLevelDataPointer levelData, ui *label, ui k, ui n, ui m, ui psize, ui cpSize, ui maxBitMask, ui level, ui totalWarps) {
    extern __shared__ char sharedMemory[];
    ui sizeOffset = 0;

    ui *counter = (ui *)(sharedMemory + sizeOffset);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;
    int cliquePartition = warpId * psize;
    int offsetPartition = warpId * (psize / (k-1) + 1);
    int candidatePartition = warpId * cpSize;
    int maskPartition = warpId * cpSize * maxBitMask;


    for(int i = warpId; i < n; i += totalWarps) {

        ui vertex = i;
        ui neighOffset = D.offset[vertex];
        if(laneId == 0) {
            counter[threadIdx.x / warpSize] = 0;
        }

        __syncwarp();

        int candidateOffset = candidatePartition + levelData.offsetPartition[offsetPartition + levelData.count[warpId + 1]];


        for(int j = laneId; j < D.degree[vertex]; j += warpSize) {
            ui neigh = D.neighbors[neighOffset + j];

            if(label[warpId*n + neigh] == k) {
                label[warpId*n + neigh] = k - 1;
                ui loc = atomicAdd(&counter[threadIdx.x / warpSize], 1);
                levelData.candidatesPartition[candidateOffset + loc] = neigh;
            }
        }
        __syncwarp();
        if(laneId == 0 && counter[threadIdx.x / warpSize] > 0) {
            levelData.partialCliquesPartition[cliquePartition + levelData.count[warpId + 1] * (k-1) + level] = vertex;
            levelData.count[warpId + 1] += 1;
            levelData.offsetPartition[offsetPartition + levelData.count[warpId + 1]] =
                levelData.offsetPartition[offsetPartition + levelData.count[warpId + 1] - 1] + counter[threadIdx.x / warpSize];
        }
         __syncwarp();

        int start = candidateOffset;

        for(int j = laneId; j < counter[threadIdx.x / warpSize]; j += warpSize) {
            int candidate = levelData.candidatesPartition[start + j];
            int neighOffset = D.offset[candidate];
            int degree = D.degree[candidate];

            int numBitmasks = (degree + 31) / 32;

            for (int bitmaskIndex = 0; bitmaskIndex < numBitmasks; bitmaskIndex++) {
                ui bitmask = 0; // Initialize bitmask to 0

                // Iterate over the current chunk of 32 neighbors
                int startNeighbor = bitmaskIndex * 32;
                int endNeighbor = min(startNeighbor + 32, degree);
                for (int x = startNeighbor; x < endNeighbor; x++) {


                    if (label[warpId*n + D.neighbors[neighOffset + x]] == k - 1) {
                        bitmask |= (1 << (x - startNeighbor)); // Set the bit for valid neighbors


                    }
                }



                levelData.validNeighMaskPartition[maskPartition + (levelData.offsetPartition[offsetPartition + levelData.count[warpId + 1]-1]+j) * maxBitMask + bitmaskIndex] = bitmask;
            }
        }

        __syncwarp();

        for(int i = laneId; i<n;i+=warpSize){
          label[warpId*n + i] = k;
        }

       __syncwarp();
    }

}

__global__ void flushParitions(deviceDAGpointer D, cliqueLevelDataPointer levelData, ui pSize, ui cpSize, ui k, ui maxBitMask, ui level, ui totalWarps){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;
    int cliquePartition = warpId * pSize;
    int offsetPartition = warpId * (pSize / (k-1) + 1);
    int candidatePartition = warpId * cpSize;
    int maskPartition = warpId * cpSize * maxBitMask;

    int totalTasks = levelData.count[warpId+1] - levelData.count[warpId];

    for(int iter = 0; iter < totalTasks; iter++){
        int start = candidatePartition + levelData.offsetPartition[offsetPartition + iter];
        int end = candidatePartition + levelData.offsetPartition[offsetPartition + iter+ 1];
        int total = end-start;

        int writeOffset = levelData.temp[warpId] + levelData.offsetPartition[offsetPartition + iter];
        for(int i = laneId; i < total; i+=warpSize){
           ui candidate = levelData.candidatesPartition[start + i];
            levelData.candidates[writeOffset+ i] = levelData.candidatesPartition[start + i];
            
            int totalMasks = (D.degree[candidate]+31)/32;
            for(int j =0; j < totalMasks; j++){
                levelData.validNeighMask[(writeOffset+i)*maxBitMask + j ] =
                levelData.validNeighMaskPartition[maskPartition + (levelData.offsetPartition[offsetPartition + iter] + i)*maxBitMask + j];
            }

        }

        if(laneId< level+1 ){

                levelData.partialCliques[levelData.count[warpId]*(k-1)+ iter*(k-1) + laneId] = levelData.partialCliquesPartition[cliquePartition + iter * (k-1) + laneId];
          }

        __syncwarp();

        if(laneId==0){

            levelData.offset[levelData.count[warpId] + iter + 1] = levelData.temp[warpId]+levelData.offsetPartition[offsetPartition + iter+ 1];

        }
        __syncwarp();


    }

}

__global__ void listMidCliques(deviceDAGpointer D, cliqueLevelDataPointer levelData, ui *label, ui k,ui iterK, ui n, ui m,ui pSize, ui cpSize, ui maxBitMask,ui totalTasks, ui level, ui totalWarps){

    extern __shared__ char sharedMemory[];
    ui sizeOffset = 0;

    ui *counter = (ui * )(sharedMemory + sizeOffset);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;
    int cliquePartition  = warpId*pSize;
    int offsetPartition = warpId*(pSize/(k-1)+1);
    int candidatePartition = warpId*cpSize;
    int maskPartition = warpId*cpSize*maxBitMask;


    for(int i =warpId; i < totalTasks ; i+= totalWarps ){

        int start = levelData.offset[i];
        int totalCandidates = levelData.offset[i+1]- start;

        for(int iter = 0; iter <totalCandidates; iter ++){
            int candidate = levelData.candidates[start + iter];
            if(laneId==0){
                counter[threadIdx.x/warpSize] = 0;
            }
            __syncwarp();

            int degree = D.degree[candidate];
            int neighOffset = D.offset[candidate];

            int writeOffset = candidatePartition + levelData.offsetPartition[offsetPartition + levelData.count[warpId + 1]];
            for(int j = laneId; j< degree; j+= warpSize ){
                int iterBitMask = j/warpSize;
                int bitPos = j%32;
                int neighBitMask = levelData.validNeighMask[start*maxBitMask + iter + iterBitMask];
                ui neigh = D.neighbors[neighOffset + j];
                if( (label[warpId*n + neigh] == iterK) && (neighBitMask & (1 << bitPos )) ){
                    label[warpId*n + neigh] = iterK-1;
                    ui loc = atomicAdd( &counter[threadIdx.x/warpSize], 1);

                    levelData.candidatesPartition[writeOffset + loc] = neigh;

                }

            }
            __syncwarp();
            if(laneId == 0 && counter[threadIdx.x/warpSize] > 0){
                levelData.partialCliquesPartition[cliquePartition + levelData.count[warpId+1] * (k-1) + level ] = candidate;
                for(int l =0; l<level; l++){
                  levelData.partialCliquesPartition[cliquePartition + levelData.count[warpId+1] * (k-1) + l ] = levelData.partialCliques[i*(k-1)+l];
                }
                levelData.count[warpId+1] +=1;
                levelData.offsetPartition[offsetPartition + levelData.count[warpId+1]] =
                    levelData.offsetPartition[offsetPartition + levelData.count[warpId+1] - 1] +counter[threadIdx.x/warpSize];
            }

            __syncwarp();
            int start = writeOffset;

            for(int j = laneId; j < counter[threadIdx.x / warpSize]; j += warpSize) {
                int cand = levelData.candidatesPartition[start + j];
                int neighOffset = D.offset[cand];
                int degree = D.degree[cand];
    
                int numBitmasks = (degree + 31) / 32;
    
                for (int bitmaskIndex = 0; bitmaskIndex < numBitmasks; bitmaskIndex++) {
                    ui bitmask = 0; // Initialize bitmask to 0
    
                    // Iterate over the current chunk of 32 neighbors
                    int startNeighbor = bitmaskIndex * 32;
                    int endNeighbor = min(startNeighbor + 32, degree);
                    for (int x = startNeighbor; x < endNeighbor; x++) {
    
    
                        if (label[warpId*n + D.neighbors[neighOffset + x]] == iterK - 1) {
                            bitmask |= (1 << (x - startNeighbor)); // Set the bit for valid neighbors
    
    
                        }
                    }
    
    
    
                    levelData.validNeighMaskPartition[maskPartition + (levelData.offsetPartition[offsetPartition + levelData.count[warpId + 1]-1]+j) * maxBitMask + bitmaskIndex] = bitmask;
                }
            }
    
            __syncwarp();
    
            for(int i = laneId; i<n;i+=warpSize){
              label[warpId*n + i] = iterK;
            }
    
           __syncwarp();
        }

    }
    
}

__global__ void writeFinalCliques(deviceGraphPointers G, deviceDAGpointer D, cliqueLevelDataPointer levelData, deviceCliquesPointer cliqueData, ui *globalCounter,ui k,ui iterK, ui n, ui m,ui pSize, ui cpSize, ui maxBitMask,ui trieSize,ui totalTasks, ui level, ui totalWarps){
    extern __shared__ char sharedMemory[];
    ui sizeOffset = 0;

    ui *counter = (ui * )(sharedMemory + sizeOffset);
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    
    for(int i =warpId; i < totalTasks ; i+= totalWarps ){

        int start = levelData.offset[i];
        int totalCandidates = levelData.offset[i+1]- start;

        for(int iter = 0; iter <totalCandidates; iter ++){
            int candidate = levelData.candidates[start + iter];
            if(laneId==0){
                counter[warpId]=0;
            }
            __syncwarp();
            int degree = D.degree[candidate];
            int neighOffset = D.offset[candidate];

            
            for(int j = laneId; j< degree; j+= warpSize ){
                int iterBitMask = j/warpSize;
                int bitPos = j%32;
                int neighBitMask = levelData.validNeighMask[start*maxBitMask + iter + iterBitMask];
                if(neighBitMask & (1 << bitPos )){

                    ui neigh = D.neighbors[neighOffset + j];
                  
                    ui loc = atomicAdd(globalCounter,1);
                    for(int ind =0; ind < k-2; ind++){
                        cliqueData.trie[trieSize * ind + loc] = levelData.partialCliques[(i)*(k-1) + ind];
                        
                    }
                    atomicAdd(&counter[warpId],1);
                    cliqueData.trie[trieSize * (k-2) + loc]  = candidate;
                    cliqueData.trie[trieSize * (k-1) + loc] = neigh;
                    cliqueData.status[loc]=-1;
                    atomicAdd(&G.cliqueDegree[neigh],1);
                    atomicAdd(&G.cliqueDegree[candidate],1);

                }
                
    
            }
            __syncwarp();

            for(int j = laneId; j< k-2 ; j+= warpSize ){
                int pClique = levelData.partialCliques[i*(k-1) + j];
                atomicAdd(&G.cliqueDegree[pClique],counter[warpId]);
            }

        }        
    }

}

__global__ void sortTrieData(deviceGraphPointers G, deviceCliquesPointer cliqueData, ui totalCliques, ui t, ui k, ui totalThreads){
    extern __shared__ char sharedMemory[];
    ui sizeOffset = 0;

    ui *elements = (ui * )(sharedMemory + sizeOffset);
    sizeOffset = k*WARPS_EACH_BLK*sizeof(ui);
    ui *degree = (ui * )(sharedMemory + sizeOffset);


    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    for(int i = idx; i <totalCliques; i+=totalThreads ){

        for(int j=0;j<k;j++){
            
            elements[j] = cliqueData.trie[j*t+i];
            degree[j] = G.cliqueDegree[elements[j]];
        }

        __syncwarp();

        // Use insertion sort, as it is best for small arrays 

        for(int j=1;j<k;j++){
            ui current_element = elements[j];
            ui current_degree = degree[j];
            int ind = j-1;

            while(ind >= 0 && degree[ind] > current_degree){
                elements[ind + 1] = elements[ind];
                degree[ind + 1] = degree[ind];
                ind--;
            }
            elements[ind + 1] = current_element;
            degree[ind + 1] = current_degree;

        }

        for(int j=0;j<k;j++){
            cliqueData.trie[j*t+i] = elements[j];
    
        }

    }

}

__global__ void selectNodes(deviceGraphPointers G, ui *bufTails,ui *glBuffers, ui glBufferSize, ui n, ui level){
    __shared__ ui *glBuffer;
    __shared__ ui bufTail;

    if(threadIdx.x == 0){
        bufTail = 0;
        glBuffer = glBuffers + blockIdx.x*glBufferSize;
    }
    __syncthreads();

    ui idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(ui i = idx ;i<n; i+=BLK_DIM){
      ui v = i;

      if(G.cliqueCore[v] == level){
        ui loc = atomicAdd(&bufTail, 1);
        glBuffer[loc] = v;
        }
    }
    __syncthreads();

    if(threadIdx.x == 0)
    {
        bufTails [blockIdx.x] = bufTail;
    }

}

__global__ void processNodesByWarp(deviceGraphPointers G,deviceCliquesPointer cliqueData, ui *bufTails,ui *glBuffers, ui *globalCount, ui glBufferSize, ui n, ui level, ui k, ui t, ui tt){
    __shared__ ui bufTail;
    __shared__ ui *glBuffer;
    __shared__ ui base;
    ui warpId = threadIdx.x / 32;
    ui laneId = threadIdx.x % 32;
    ui regTail;
    ui i;
    if(threadIdx.x==0){
    bufTail = bufTails[blockIdx.x];
    base = 0;
    glBuffer = glBuffers + blockIdx.x*glBufferSize;
    assert(glBuffer!=NULL);
    }

    while(true){
    __syncthreads();
    if(base == bufTail) break; // all the threads will evaluate to true at same iteration
    i = base + warpId;
    regTail = bufTail;
    __syncthreads();

    if(i >= regTail) continue; // this warp won't have to do anything

    if(threadIdx.x == 0){
    base += WARPS_EACH_BLK;
    if(regTail < base )
    base = regTail;
    }
    //bufTail is incremented in the code below:
    ui v = glBuffer[i];


   __syncwarp();
    for(ui j =laneId; j<tt; j+=warpSize){
    //printf("warpId %d laneId %u vertex %u check %d t %d \n",warpId,i,v,cliqueData.trie[j],t);
        if(cliqueData.status[j] == -1){
            bool found = false;
            ui w =0;
            while(w<k){
                if(cliqueData.trie[w*t+j] == v){
                found = true;
                break;
                }
                w++;
            }
            if(found){

                for(ui x =0;x<k;x++){

                    if(x==w) continue;
                    ui u = cliqueData.trie[x*t+j];
                    int a = atomicSub(&G.cliqueCore[u],1);
                    if(a == level+1){
                        ui loc = atomicAdd(&bufTail, 1);
                        glBuffer[loc] = u;

                    }
                    if(a <= level){
                        atomicAdd(&G.cliqueCore[u], 1);
                    }

                }
                cliqueData.status[j] = level;


            }
        }
    }

    __syncwarp();
    if(laneId == 0 && bufTail>0){
      atomicAdd(globalCount, 1); // atomic since contention among blocks
    }
}
}

__global__ void processNodesByBlock(deviceGraphPointers G,deviceCliquesPointer cliqueData, ui *bufTails,ui *glBuffers, ui *globalCount, ui glBufferSize, ui n, ui level, ui k, ui t, ui tt){
    __shared__ ui bufTail;
    __shared__ ui *glBuffer;
    __shared__ ui base;

    ui regTail;
    ui i;
    if(threadIdx.x==0){
    bufTail = bufTails[blockIdx.x];
    base = 0;
    glBuffer = glBuffers + blockIdx.x*glBufferSize;
    assert(glBuffer!=NULL);
    }



    while(true){
        __syncthreads();
        if(base == bufTail) break; // all the threads will evaluate to true at same iteration
        i = base + blockIdx.x;
        regTail = bufTail;
        __syncthreads();

        if(i >= regTail) continue; // this warp won't have to do anything

        if(threadIdx.x == 0){
            base += 1;
            if(regTail < base )
            base = regTail;
        }
        //bufTail is incremented in the code below:
        ui v = glBuffer[i];

        __syncthreads();
        ui idx = threadIdx.x;

        for(ui j = idx; j<tt; j+= BLK_DIM){
            
            if(cliqueData.status[j]==-1){

                bool found = false;
                ui w =0;
                while(w<k){
                    if(cliqueData.trie[w*t+j] == v){
                    found = true;
                    break;
                    }
                    w++;
                }

                if(found){
                    for(ui x =0;x<k;x++){
                        if(x==w) continue;

                        ui u = cliqueData.trie[x*t+j];
                        int a = atomicSub(&G.cliqueCore[u], 1);
                        if(a == level+1){
                            ui loc = atomicAdd(&bufTail, 1);
                            glBuffer[loc] = u;

                        }
                        if(a <= level){
                            atomicAdd(&G.cliqueCore[u], 1);
                        }
                    }
                    cliqueData.status[j] = level;


                }
            }
        }


        __syncthreads();

        if(threadIdx.x == 0 && bufTail>0){
            atomicAdd(globalCount, 1); // atomic since contention among blocks
        }
    }
}

__global__ void generateDensestCore(deviceGraphPointers G, densestCorePointer densestCore,ui *globalCount, ui n, ui maxDensityCore, ui totalWarps){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < n; i += totalWarps){
        if(G.cliqueCore[i]>= maxDensityCore){
            ui loc;
            if(laneId==0){
                loc = atomicAdd(globalCount,1);
                densestCore.mapping[loc] = i;
            }
            loc = __shfl_sync(0xFFFFFFFF, loc, 0, 32);
            ui start = G.offset[i];
            ui end = G.offset[i+1];
            ui total = end - start;
            ui neigh;
            int count = 0;
            for(int j = laneId; j < total; j += warpSize) {
                neigh = G.neighbors[start + j];
                if(G.cliqueCore[neigh] >= maxDensityCore) {
                    count++;
                }
            }
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                count += __shfl_down_sync(0xFFFFFFFF, count, offset);
            }
            if(laneId == 0) {
                densestCore.offset[loc+1] = count;
            }

        }
    }
}

__global__ void generateNeighborDensestCore(deviceGraphPointers G, densestCorePointer densestCore, ui maxDensityCore, ui totalWarps) {

    extern __shared__ char sharedMemory[];
    ui sizeOffset = 0;

    ui *counter = (ui *)(sharedMemory + sizeOffset);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < (*densestCore.n); i += totalWarps) {
        if(laneId==0){
          counter[threadIdx.x / warpSize] = densestCore.offset[i];
        }
        __syncwarp();
        ui vertex = densestCore.mapping[i];
        ui start = G.offset[vertex];
        ui end = G.offset[vertex+1];
        ui total = end - start;
        ui neigh;
        for(int j = laneId; j < total; j += warpSize) {
            neigh = G.neighbors[start + j];

            if(G.cliqueCore[neigh] >= maxDensityCore) {
                int loc = atomicAdd(&counter[threadIdx.x / warpSize], 1);

                densestCore.neighbors[loc] = densestCore.reverseMap[neigh];

            }
        }
      __syncwarp();
    }
}

__global__ void pruneEdges(densestCorePointer densestCore, deviceCliquesPointer cliqueData, ui *pruneStatus,ui t, ui tt, ui k, ui level ){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i<tt; i+=TOTAL_WARPS){

        if(cliqueData.status[i] >= level){

            for(ui iter =0; iter< k ; iter ++){
                // v should be mapped
                ui u_ = ((iter)%k)*t+i;
                ui u  =  densestCore.reverseMap[cliqueData.trie[u_]];
                for(ui j = 0; j < k; j++){
                    ui v_ = ((j)%k)*t+i;


                    if(v_!=u_){
                        int v = densestCore.reverseMap[cliqueData.trie[v_]];


                        // Update u-v edge status
                        ui start = densestCore.offset[u];
                        ui end = densestCore.offset[u+1];
                        ui total = end-start;

                        for(ui ind = laneId; ind < total; ind +=WARPSIZE){
                            int neigh = densestCore.neighbors[start+ind];
                            if(neigh == v){
                              atomicCAS(&pruneStatus[start+ind], 1, 0);
                            }

                        }

                    }
                }




        }

        }
    }
}

__global__ void generateDegreeAfterPrune(densestCorePointer densestCore ,ui *pruneStatus, ui *newOffset, ui n, ui m, ui totalWarps) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < n; i += totalWarps) {
        ui start = densestCore.offset[i];
        ui end = densestCore.offset[i+1];
        ui total = end - start;
        int count = 0;
        for(int j = laneId; j < total; j += warpSize) {
            if(!pruneStatus[start + j]) {
                count++;
            }
        }

        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            count += __shfl_down_sync(0xFFFFFFFF, count, offset);
        }

        if(laneId == 0) {
            newOffset[i+1] = count;
        }
    }
}

__global__ void generateNeighborAfterPrune(densestCorePointer densestCore ,ui *pruneStatus, ui *newOffset, ui *newNeighbors,ui n, ui m, ui totalWarps) {

    extern __shared__ char sharedMemory[];
    ui sizeOffset = 0;

    ui *counter = (ui *)(sharedMemory + sizeOffset);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < n; i += totalWarps) {
        if(laneId==0){
          counter[threadIdx.x / warpSize] = newOffset[i];
        }
        __syncwarp();
        ui start = densestCore.offset[i];
        ui end = densestCore.offset[i+1];
        ui total = end - start;
        ui neigh;
        for(int j = laneId; j < total; j += warpSize) {
            neigh = densestCore.neighbors[start + j];

            if(!pruneStatus[start + j]) {
                int loc = atomicAdd(&counter[threadIdx.x / warpSize], 1);
                newNeighbors[loc] = neigh;

            }
        }
      __syncwarp();
    }
}

__global__ void componentDecomposek(deviceComponentPointers conComp, devicePrunedNeighbors prunedNeighbors, ui *changed, ui n, ui m, ui totalWarps) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;
    bool threadChanged = false;

    for(ui i = warpId; i < n; i += totalWarps) {
        ui currentComp = conComp.components[i];
        ui start = prunedNeighbors.newOffset[i];
        ui end = prunedNeighbors.newOffset[i+1];
        ui total = end - start;
        //printf("warpid %d laneId %d start %d end %d total %d cc %d \n",warpId,laneId,start,end,total,currentComp);

        ui minNeighComp = currentComp;

        for (ui j = laneId; j < total; j += warpSize) {
            ui neighComp = conComp.components[prunedNeighbors.newNeighbors[start+j]];
            minNeighComp = min(minNeighComp, neighComp);
            //printf("warp Id %d laneid %d cc %d nc %d mc %d \n",warpId,laneId,currentComp,neighComp,minNeighComp);
        }

        for (int offset = warpSize/2; offset > 0; offset /= 2) {
            ui temp = __shfl_down_sync(0xFFFFFFFF, minNeighComp, offset);
            minNeighComp = min(minNeighComp, temp);
        }

        if (laneId == 0) {
            if ( minNeighComp < currentComp) {
                conComp.components[i] = minNeighComp;
                threadChanged = true;
            }
        }

        __syncwarp();
    }

    bool warpChanged = __any_sync(0xFFFFFFFF, threadChanged);
    if (warpChanged && laneId == 0) {
        atomicAdd(changed, 1);
    }
}

__global__ void getConnectedComponentStatus(deviceComponentPointers conComp,deviceCliquesPointer cliqueData, densestCorePointer densestCore, ui *compCounter, ui t, ui tt, ui k,ui maxCore, ui totalThreads){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(ui i =idx; i<tt;i +=totalThreads){
        if(cliqueData.status[i]>=maxCore){
          int comp = INT_MAX;

          for(ui x=0;x<k;x++){
            ui vertex =densestCore.reverseMap[cliqueData.trie[x*t + i]];
            comp = min(comp,conComp.components[vertex]);

            cliqueData.trie[x*t + i] = vertex;


          }
          cliqueData.status[i] = comp;
          atomicAdd(&compCounter[comp+1],1);

        
      }else{
        cliqueData.status[i] = -1;
      }
    }

}

__global__ void rearrangeCliqueData(deviceComponentPointers conComp,deviceCliquesPointer cliqueData, deviceCliquesPointer finalCliqueData,densestCorePointer densestCore, ui *compCounter,ui *counter,ui t, ui tt, ui k,ui totaLCliques, ui totalThreads){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for(ui i =idx; i<tt;i +=totalThreads){
        
        int comp =  cliqueData.status[i];

        if(comp>-1){
          ui loc; 
          for(ui j=0;j<k;j++){
              ui vertex = cliqueData.trie[j*t + i];
              ui offset = compCounter[comp];
              if(j==0){
                  loc = atomicAdd(&counter[comp],1);
                  finalCliqueData.status[offset + loc ] = comp;


              }
              finalCliqueData.trie[offset + j*totaLCliques + loc ] = vertex;
          }
        }
        
    }

}

 __global__ void getLbUbandSize(deviceComponentPointers conComp, ui *compCounter, double *lowerBound, double *upperBound, ui *ccOffset,  ui *neighborSize, ui totalComponenets, ui k, double maxDensity){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx==0){
        ccOffset[idx]=0;
        neighborSize[idx]=0;

    }

    for(ui i = idx; i<totalComponenets; i+=TOTAL_THREAD ){
        ui totalCliques = compCounter[i+1] - compCounter[i];
        ui totalSize = conComp.componentOffset[i+1] -  conComp.componentOffset[i];
        double lb = (double) (totalCliques)/totalSize;
        lowerBound[i]  = lb;

        double dem = pow(fact(k),1.0/k);
        double num = pow(totalCliques, (k-1.0)/k);
        double ub = min(maxDensity, num/dem);

        upperBound[i] = ub;

        if(ub>lb){
            ccOffset[i+1] = totalCliques + totalSize + 2 +1 ;
            neighborSize[i+1] = 2*(2*totalCliques*k + 2*totalSize)+1;


        }
        else{
            ccOffset[i+1] = 0;
            neighborSize[i+1] = 0;

        }
    }


}


__global__ void createFlowNetworkOffset(deviceGraphPointers G, deviceFlowNetworkPointers flowNetwork, deviceComponentPointers conComp, densestCorePointer densestCore, deviceCliquesPointer finalCliqueData, ui *compCounter,double *upperBound , ui totalWarps, ui totalComponents, ui k, double lb, ui t){



    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < totalComponents; i += totalWarps){

        if(upperBound[i]>lb){

            ui start = conComp.componentOffset[i];
            ui end = conComp.componentOffset[i+1];
            ui total = end - start;
            ui startClique = compCounter[i];
            ui totalCliques = compCounter[i+1]-compCounter[i];

            ui vertexOffset = flowNetwork.offset[i];
            //ui neighborOffset = flowNetwork.neighborOffset1[i];

            //printf("warpid %d laneId %d start %d end %d total %d neighborOffset %d \n",warpId,laneId,start,end,total,neighborOffset);

            for (ui j = laneId; j < total; j += warpSize){
                ui vertex = conComp.mapping[start+j];

                ui cliqueDegree = 0;
                for(ui x =0; x < totalCliques; x ++){
                  ui u;
                  for(ui k_ = 0; k_<k;k_++){
                    u = finalCliqueData.trie[t*k_ + startClique+x];
                    if (u==vertex){
                      cliqueDegree++;
                      }
                  }


                }
                flowNetwork.neighborOffset2[vertexOffset+j + 1] = 2*(cliqueDegree + 1);
            }
            for (ui j = laneId; j < totalCliques; j += warpSize){
                flowNetwork.neighborOffset2[vertexOffset+total+j+1] = 2*k;
            }
            if(laneId==0){
                flowNetwork.neighborOffset2[vertexOffset+total+totalCliques+1] = total;
                flowNetwork.neighborOffset2[vertexOffset+total+totalCliques+2] = total;
                flowNetwork.neighborOffset2[0]=0;
            }


            }
        }
    }

__global__ void createFlowNetwork(deviceFlowNetworkPointers flowNetwork, deviceComponentPointers conComp, densestCorePointer densestCore, deviceCliquesPointer finalCliqueData, ui *compCounter,double *upperBound , ui totalWarps, ui totalComponents, ui k, double lb, ui t){



    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;

    for(ui i = warpId; i < totalComponents; i += totalWarps){

        if(upperBound[i]>lb){

            ui start = conComp.componentOffset[i];
            ui end = conComp.componentOffset[i+1];
            ui total = end - start;
            ui startClique = compCounter[i];
            ui totalCliques = compCounter[i+1]-compCounter[i];

            ui vertexOffset = flowNetwork.offset[i];
            ui neighborOffset1 = flowNetwork.neighborOffset1[i];

            double alpha = upperBound[i]/lb;

            //printf("warpid %d laneId %d start %d end %d total %d neighborOffset %d \n",warpId,laneId,start,end,total,neighborOffset);

            for (ui j = laneId; j < total; j += warpSize){
                ui neighborOffset = flowNetwork.neighborOffset2[vertexOffset+j];


                // Vertex to sink
                flowNetwork.Edges[neighborOffset] = total+totalCliques+1;
                flowNetwork.capacity[neighborOffset] = alpha * k;

                //vertex to source (backward)
                flowNetwork.Edges[neighborOffset+1] = total+totalCliques;
                flowNetwork.capacity[neighborOffset+1] = 0.0;


                ui vertex = conComp.mapping[start+j];

                ui cliqueDegree = flowNetwork.neighborOffset2[vertexOffset+j + 1] - flowNetwork.neighborOffset2[vertexOffset+j];


                ui temp= 2;

                for(ui x =0; x < totalCliques; x ++){
                    ui u;
                    for(ui k_ = 0; k_<k;k_++){
                      u = finalCliqueData.trie[t*k_ + startClique+x];
                      if (u==vertex){
                        //vertex to clique
                        flowNetwork.Edges[neighborOffset+ temp] = total+x;
                        flowNetwork.capacity[neighborOffset+ temp] = 1.0;

                        // vertex to clique backward
                        flowNetwork.Edges[neighborOffset+ temp+ 1] = total+x;
                        flowNetwork.capacity[neighborOffset+ temp+1] = 0.0;
                        temp+=2;
                        }
                    }
                    if(temp==(2*cliqueDegree)){
                        break;
                    }


                  }

            }
            for (ui j = laneId; j < totalCliques; j += warpSize){
                ui neighborOffset = flowNetwork.neighborOffset2[vertexOffset+total+j];
                ui u;
                for(ui k_ = 0; k_<k;k_++){
                    u = finalCliqueData.trie[t*k_ + startClique+j];

                    //Clique to vertex
                    flowNetwork.Edges[neighborOffset+ 2*k_ ] = conComp.reverseMapping[u] -start;
                    flowNetwork.capacity[neighborOffset+ 2*k_] = DINF;

                    //Clique to vertex backward
                    flowNetwork.Edges[neighborOffset+ 2*k_ +1 ] = conComp.reverseMapping[u] -start;
                    flowNetwork.capacity[neighborOffset+ 2*k_ + 1] = 0;

                }


            }
            ui neighborOffset_source = flowNetwork.neighborOffset2[vertexOffset+total+totalCliques];
            ui neighborOffset_sink = flowNetwork.neighborOffset2[vertexOffset+total+totalCliques +1 ];



            for (ui j = laneId; j < total; j += warpSize){

                ui cliqueDegree = (flowNetwork.neighborOffset2[vertexOffset+j + 1] - flowNetwork.neighborOffset2[vertexOffset+j]-2)/2;

                //source to vertex
                flowNetwork.Edges[neighborOffset_source+ j] = j;
                flowNetwork.capacity[neighborOffset_source+ j] = (double) cliqueDegree;


                //sink to vertex backward
                 flowNetwork.Edges[neighborOffset_sink+ j] = j;
                flowNetwork.capacity[neighborOffset_sink+ j] = 0.0;


            }



            }

    }
 }

__global__ void pushRelabel(deviceFlowNetworkPointers flowNetwork, deviceComponentPointers conComp, densestCorePointer densestCore, deviceCliquesPointer finalCliqueData, ui * compCounter, double * upperBound, double * lowerBound, ui * activeNodes, ui * componenetsLeft, ui totalWarps, int totalComponents, ui k,ui t, ui partitionSize) {
  extern __shared__ char sharedMemory[];
  ui sizeOffset = 0;

  ui * counter = (ui * )(sharedMemory + sizeOffset);

  // += WARPS_EACH_BLK * sizeof(ui);
  //sizeOffset = (sizeOffset + alignof(double) - 1) & ~(alignof(double) - 1);
  //ui *densities = (ui *)(sharedMemory + sizeOffset);

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int warpId = idx / warpSize;
  int laneId = idx % warpSize;

  for (ui i = warpId; i < totalComponents; i += totalWarps) {
    ui start = conComp.componentOffset[i];
    ui end = conComp.componentOffset[i + 1];
    ui total = end - start;

    ui cliqueStart = compCounter[i];
    ui cliqueEnd = compCounter[i + 1];
    ui totalCliques = cliqueEnd - cliqueStart;

    ui fStart = flowNetwork.offset[i];

    ui tFlow = totalCliques + total + 2;

    double bais = 1.0 / (tFlow * (tFlow - 1));

    if ((upperBound[i] - lowerBound[i]) > bais) {
      

      ui s = tFlow - 2;
      ui t = tFlow - 1;

      //Set Height to 0 expect for s to total
      for (ui j = laneId; j < tFlow; j += warpSize) {
        flowNetwork.height[fStart + j] = (j == s) ? tFlow : 0;
        flowNetwork.excess[fStart + j] = 0;

      }
      __syncwarp();

      // Send Intial flow for s to all v
      for (ui j = laneId; j < total; j += warpSize) {
        ui nStart = flowNetwork.neighborOffset2[fStart + totalCliques + total];
        ui neigh = flowNetwork.Edges[nStart + j];
        double cap = flowNetwork.capacity[nStart + j];

        //Forward Flow s to vertex
        flowNetwork.flow[nStart + j] = cap;
        atomicAdd( & flowNetwork.excess[fStart + neigh], cap);

        //BackwardFlow vertex to s
        flowNetwork.flow[flowNetwork.neighborOffset2[fStart + neigh] + 1] = -cap;

      }

      __syncwarp();

      int maxIterations = 4;

      //printf("warpid %d laneid %d\n ", warpId,laneId);

      //Push or Relabel until converge
      for (int iter = 0; iter < maxIterations; iter++) {
        if (laneId == 0) {
          counter[threadIdx.x / warpSize] = 0;
        }
        __syncwarp();

        //check for nodes with excess
        for (ui j = laneId; j < tFlow; j += warpSize) {
          if (j != s && j != t && flowNetwork.excess[fStart + j] > 0) {
            int pos = atomicAdd( & counter[threadIdx.x / warpSize], 1);
            if (pos < partitionSize) {
              activeNodes[i * partitionSize + pos] = j;
            }
           // printf("warp id %d lane id %d pos %d u %d  iter %d \n", i, j, pos, j, iter);

          }
        }
        __syncwarp();
        bool should_break = (counter[threadIdx.x / warpSize] == 0);
        __syncwarp();  // Sync before any thread exits

        if (should_break) break;

        bool pushed = false;

        for (ui j = laneId; j < counter[threadIdx.x / warpSize]; j += warpSize) {
          ui vertex = activeNodes[i * partitionSize + j];

          ui nStart = flowNetwork.neighborOffset2[fStart + vertex];
          ui nEnd = flowNetwork.neighborOffset2[fStart + vertex + 1];
          //printf("warpid %d lane id %d read ver %d excess %f height %d offset %d nstart %d end %d \n", i, j, vertex, flowNetwork.excess[fStart + vertex], flowNetwork.height[fStart + vertex], fStart, nStart, nEnd);

          //Check neighbors to send acess to.
          for (ui x = nStart; x < nEnd; x++) {
            //
            /*if (flowNetwork.excess[fStart + vertex] == 0) {
              break;
            }*/
            ui neigh = flowNetwork.Edges[x];
            double residual = flowNetwork.capacity[x] - flowNetwork.flow[x];


            __syncwarp();

            // If neighbor has capacity
            if ((flowNetwork.height[fStart + vertex] == flowNetwork.height[fStart + neigh] + 1) && residual > 0) {

              //printf("warp Id %d laneid %d vertex %d nstart %d end %d neigh %d resi %f cap %f flow %f loc %d hv %d hu %d ev %f en %f \n", i, j, vertex, nStart, nEnd, neigh, residual, flowNetwork.capacity[x], flowNetwork.flow[x], x, flowNetwork.height[fStart + vertex], flowNetwork.height[fStart + neigh], flowNetwork.excess[fStart + vertex], flowNetwork.excess[fStart + neigh]);

              double delta = min(flowNetwork.excess[fStart + vertex], residual);
              if (delta > 0) {

                //forward flow vertex to neigh
                atomicAdd( & flowNetwork.flow[x], delta);

                //Backward Flow neigh to vertex
                ui stemp = flowNetwork.neighborOffset2[fStart + neigh];
                ui etemp = flowNetwork.neighborOffset2[fStart + neigh + 1];
                for (ui ind = stemp; ind < etemp; ind++) {
                  if (flowNetwork.Edges[ind] == vertex) {
                    atomicAdd( & flowNetwork.flow[ind], -delta);
                    break;
                  }
                }

                //Decrease vertex excess
                atomicAdd( & flowNetwork.excess[fStart + vertex], -delta);

                //Increase neighbor excess
                atomicAdd( & flowNetwork.excess[fStart + neigh], delta);
                pushed = true;

              }
             // printf("after -- warp Id %d laneid %d vertex %d nstart %d end %d neigh %d resi %f cap %f flow %f loc %d hv %d hu %d ev %f en %f \n", i, j, vertex, nStart, nEnd, neigh, residual, flowNetwork.capacity[x], flowNetwork.flow[x], x, flowNetwork.height[fStart + vertex], flowNetwork.height[fStart + neigh], flowNetwork.excess[fStart + vertex], flowNetwork.excess[fStart + neigh]);

            }

          }

          __syncwarp();

          // Relabel
          if (!pushed && flowNetwork.excess[fStart + vertex] > 0) {
            ui minHeight = UINT_MAX;
            //printf("Relabel  WarpId %d laneid %d vertex %d excess %f height %d nstart %d end %d \n", i, laneId, vertex, flowNetwork.excess[fStart + vertex], flowNetwork.height[fStart + vertex], nStart, nEnd);
            for (ui x = nStart; x < nEnd; x++) {
              ui neigh = flowNetwork.Edges[x];
              double residual = flowNetwork.capacity[x] - flowNetwork.flow[x];
              //printf("warp %d laneid %d j %d v %d res %f negh %d height %d nstart %d \n", i, laneId, j, vertex, residual, neigh, flowNetwork.height[fStart + neigh], nStart);

              if (residual > 0.0) {
                minHeight = (flowNetwork.height[fStart + neigh] < minHeight) ? flowNetwork.height[fStart + neigh] : minHeight;
              }

            }
            if (minHeight != INF) {
              flowNetwork.height[fStart + vertex] = minHeight + 1;

            }

            //printf("after Relabel  WarpId %d laneid %d vertex %d excess %f height %d minHeight %d \n", i, laneId, vertex, flowNetwork.excess[fStart + vertex], flowNetwork.height[fStart + vertex], minHeight);

          }
          __syncwarp();
        }

      }

      __syncwarp();


      //printf("second warpid %d laneid %d\n ", warpId,laneId);

      if(laneId==0){
        //getRes[threadIdx.x / warpSize] = 0;
        double alpha = (upperBound[i] + lowerBound[i]) / 2;
        if (flowNetwork.excess[fStart + t] == (double) totalCliques * k) {
            upperBound[i] = alpha;
        } else {
            lowerBound[i] = alpha;

        }
        if ((upperBound[i] - lowerBound[i]) > bais) {
            atomicAdd(componenetsLeft, 1);

        }

      }

      __syncwarp();


      //Update the network
      if ((upperBound[i] - lowerBound[i]) > bais){
            double alpha = (upperBound[i] + lowerBound[i]) / 2;
            for (ui j = laneId; j < total; j += warpSize){
                ui neighborOffset = flowNetwork.neighborOffset2[fStart+j];
                flowNetwork.capacity[neighborOffset] = alpha * k;


            }

      }

    }

  }
}

__global__ void getResult(deviceFlowNetworkPointers flowNetwork, deviceComponentPointers conComp, deviceCliquesPointer finalCliqueData, ui *compCounter,  double * upperBound, double * lowerBound, double *densities,ui totalWarps, int totalComponents, ui k,ui t) {

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int warpId = idx / warpSize;
  int laneId = idx % warpSize;

  for (ui i = warpId; i < totalComponents; i += totalWarps) {
    ui start = conComp.componentOffset[i];
    ui end = conComp.componentOffset[i + 1];
    ui total = end - start;
    ui cliqueStart = compCounter[i];
    ui cliqueEnd = compCounter[i + 1];
    ui totalCliques = cliqueEnd - cliqueStart;

    ui tFlow = total + totalCliques +2;

    double bais = 1.0 / (tFlow * (tFlow - 1));


    if (((upperBound[i] - lowerBound[i]) < bais)&&(upperBound[i]!=0)&&(lowerBound[i]!=0)) {

        ui size = 0;
        ui fStart = flowNetwork.offset[i];
        for (ui j = 0; j < total; j ++){
          ui neighborOffset = flowNetwork.neighborOffset2[fStart+total+totalCliques];
          if(laneId==0)
            printf("warpId %d lane id %d j %d o %d no %d vertex %d cap %f flow %f \n",warpId, laneId,j, fStart,neighborOffset, flowNetwork.Edges[neighborOffset+j],flowNetwork.capacity[neighborOffset+j],flowNetwork.flow[neighborOffset+j]);
             
             double residual = flowNetwork.capacity[neighborOffset+j] - flowNetwork.flow[neighborOffset+j];
             if(residual >0 ){
                size++;
                int vertex = flowNetwork.Edges[neighborOffset+j];
                for(ui x = laneId; x < totalCliques; x +=warpSize){
                    ui w = 0;
                    while(w<k){
                        if(finalCliqueData.trie[w*t+x + cliqueStart ] == vertex){
                            atomicAdd(&densities[i],1);
                        break;
                        }
                        w++;
                    }

                }
                 __syncwarp();
             }

        }
         __syncwarp();

        if(laneId==0){
            // Total Cliques
            densities[i]= densities[i]/k;

            //Density
            if(size==0){
              densities[i] = lowerBound[i];
            }else{
              densities[i]= densities[i]/size;
            }
            
            printf("Warpid %d density %f size %d \n", warpId,densities[i],size);


        }


  }
}
}




/*__global__ void edmondsKarp(deviceFlowNetworkPointers flowNetwork, deviceComponentPointers conComp, densestCorePointer densestCore, deviceCliquesPointer finalCliqueData, ui *compCounter,ui *counter,ui *upperBound, ui *ranks,ui *offset, int *augmentedPaths, ui *BFS, ui apSize, ui n, ui m, ui totalWarps, int totalComponents, ui k, ui lb){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(ui i = idx; i < totalComponents; i += TOTAL_THREAD){
        if(upperBound[i]>lb){
            ui alpha = (upperBound[i]+lb)/2;
            ui offsetLoc;
            ui start = conComp.componentOffset[i];
            ui end = conComp.componentOffset[i+1];
            ui total = end - start;

            ui bais = 1/(total*(total-1));

            ui cliqueStart = compCounter[i];
            ui cliqueEnd = compCounter[i+1];
            ui totalCliques = cliqueEnd-cliqueStart;

            ui offset = offset[rank[i]];

            ui apOffset = apSize*i;

            ui s = totalCliques + total;
            ui t = totalCliques + total + 1;

            ui curent = s;
            ui count = 1;
            ui tail = 0;
            BFS[apOffset] = s;
            ui maxFlow =INF;
            ui current;

            while( (u-l) > bais ){

                while(tail < count){
                    ui current = BFS[apOffset + tail];
                    tail++;
                    if(current == t){

                        while(current!=s){
                            ui nStart= flowNetwork.offset[offset + augmentedPaths[current]];
                            ui nEnd = flowNetwork.offset[offset + augmentedPaths[current]+1];
                            for(ui j = nStart; j < nEnd; j++ ){
                                if(j==current){
                                    ui availCap =  flowNetwork.capacity[nStart+j] -flowNetwork.flow[nStart+j];
                                    if(maxFlow > availCap){
                                        maxflow = availCap;
                                    }
                                    
                                    break;
                                }
                            }

                            current = augmentedPaths[current];


                        }

                    }

                    ui nStart= flowNetwork.offset[offset + current];
                    ui nEnd = flowNetwork.offset[offset + current+1];

                    for(ui j = nStart; j < nEnd; j++ ){
                        ui v = flowNetwork.toEdge[nStart+j];
                        ui availCap =  flowNetwork.capacity[nStart+j] -flowNetwork.flow[nStart+j];
                        if((augmentedPaths[apOffset+v] == -1) && availCap>0){
                            augmentedPaths[apOffset+v] = current;
                            BFS[apOffset+count] = v;
                            count++;
                        }
                    }

                }
                
            }


        }


    }
}*/

/*__global__ void createPaths(deviceFlowNetworkPointers flowNetwork, deviceComponentPointers conComp, ui totalWarps, int totalComponents, ui k, ui alpha){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;
    ui offset = i*(2*totalCliques*k+4*total);
    for(ui i = warpId; i < totalComponents; i += totalWarps){
        



    }

}




    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warpId = idx / warpSize;
    int laneId = idx % warpSize;
    bool threadChanged = false;

    for(ui i = warpId; i < totalComponents; i += totalWarps){
        ui startComp = conComp.componentOffset[i];;
        ui endComp = conComp..componentOffset[i+1];
        ui total = endComp-startComp;



        //Create Flow network


        //get lower bound upper bound and bais

        //create spanning tree for forward edges

        //run algo

        // run algo again with backward edges 

        //atomic max for max density 



    }

}*/